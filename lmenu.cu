#include "hip/hip_runtime.h"
/* Copyright 1995-2000 Just For Fun Software, Inc. */
/* Author:  George Woltman */
/* Email: woltman@alum.mit.edu */
/* Adapted for LLR program by Jean Penn� */
/* Email : jpenne@free.fr */

/* Include files */

//#include <string.h>
//#include <stdio.h>

#include <ctype.h>
#include <fcntl.h>
#include <math.h>
#include <memory.h>
#include <signal.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/stat.h>
#if defined (__linux__) || defined (__FreeBSD__) || defined (__APPLE__)
#include <dirent.h>
#include <unistd.h>
#include <sys/time.h>
#include <sys/resource.h>
#define $LLF "%qi"
#define __int64 long long
#else
#include <direct.h>
#include <dos.h>
#include <io.h>
#include <time.h>
#include <process.h>
#include <windows.h>
#define $LLF "%I64d"
#endif
#include <sys/timeb.h>

#if !defined (__linux__) && !defined (__FreeBSD__) && !defined (__APPLE__)

#include <io.h>
#include <stdlib.h>

#endif

#include "lprime.h"
//#include "gwypini.c"

/* Routine definitions */

void options_cpu ();

/* Get line from the user (stdin) */

void get_line (
	char	*buf)
{
	char	c;
	int	i;
	for (i = 0; ; i++) {
		if (_read (0, &c, 1) != 1) break;
		if (c == '\n' || c == 0) break;
		if (i < 80) *buf++ = c;
	}
	*buf++ = 0;
}

/* Get a number from the user */

unsigned long get_number (
	unsigned long dflt)
{
	char	line[80];
	get_line (line);
	if (line[0] == 0) return (dflt);
	return (atol (line));
}

/* Ask a Yes/No question */

void askYN (
	char	*str,
	int	*val)
{
	char	buf[80];
	printf ("%s (%s): ", str, *val ? "Y" : "N");
	get_line (buf);
	if (buf[0] == 0) return;
	*val = (buf[0] == 'Y' || buf[0] == 'y');
}

/* Ask a number question */

void askNum (
	char	*str,
	unsigned long *val,
	unsigned long min,
	unsigned long max)
{
	char	buf[80];
	unsigned long newval;
	printf ("%s (%ld): ", str, *val);
loop:	get_line (buf);
	if (buf[0] == 0) return;
	newval = atol (buf);
	if (min || max) {
		if (newval < min || newval > max) {
			printf ("Please enter a value between %ld and %ld. ",
				min, max);
			goto loop;
		}
	}
	*val = newval;
}

/* Ask a number question */

void askNumNoDflt (
	char	*str,
	unsigned long *val,
	unsigned long min,
	unsigned long max)
{
	char	buf[80];
	unsigned long newval;
	printf ("%s: ", str);
loop:	get_line (buf);
	if (buf[0] == 0) goto loop;
	newval = atol (buf);
	if (min || max) {
		if (newval < min || newval > max) {
			printf ("Please enter a value between %ld and %ld. ",
				min, max);
			goto loop;
		}
	}
	*val = newval;
}

/* Ask a string question */

void askStr (
	char	*str,
	char	*val,
	unsigned long maxlen)
{
	char	buf[80];
	if (val[0])
		printf ("%s (%s): ", str, val);
	else
		printf ("%s: ", str);
loop:	get_line (buf);
	if (buf[0] == 0) return;
	if (strlen (buf) > maxlen) {
		printf ("Maximum string length is %ld characters. ", maxlen);
		goto loop;
	}
	strcpy (val, buf);
}

/* Wait for user input - gives the user time to read the screen */

void askOK ()
{
	char	str[80];
	if (THREAD_KILL) return;
	printf ("\nHit enter to continue: ");
	get_line (str);
}

/* Ask user if he is satisfied with his dialog responses */

int askOkCancel ()
{
	char	buf[80];
	if (THREAD_KILL) return (FALSE);
	printf ("\nAccept the answers above? (Y): ");
	get_line (buf);
	return (buf[0] == 0 || buf[0] == 'Y' || buf[0] == 'y');
}

/* Ask user if he is satisfied with his dialog responses */

int askYesNo (
	char	dflt)
{
	char	buf[80];
	if (THREAD_KILL) return (FALSE);
	printf (" (%c): ", dflt);
	get_line (buf);
	if (buf[0] == 0) buf[0] = dflt;
	return (buf[0] == 'Y' || buf[0] == 'y');
}

/* Output a long string with a max of 75 characters to a line */

void outputLongLine (
	char	*buf)
{
	char	line[80];
	char	*p;
	int	i, j;

	for (p = buf; ; ) {
		for (i = 0; i < 75; i++) {
			line[i] = p[i];
			if (p[i] == 0 || p[i] == '\n') { j = i; break; }
			if (p[i] == ' ' || p[i] == '.' || p[i] == ',') j = i;
		}
		line[j+1] = 0;
		printf ("%s", line);
		if (p[j] == 0) break;
		if (p[j] != '\n') printf ("\n");
		p += j + 1;
		while (*p == ' ') p++;
	}
}

/* Test/InputData dialog */

void test_inputdata ()
{
	char	m_pgen_input[80], m_pgen_output[80];
	unsigned long m_pgen_line;

	IniGetString (INI_FILE, (char*)"PgenInputFile", m_pgen_input, 80, NULL);
	IniGetString (INI_FILE, (char*)"PgenOutputFile", m_pgen_output, 80, NULL);
	m_pgen_line = IniGetInt (INI_FILE, (char*)"PgenLine", 1);

	askStr ((char*)"Input file (from NewPgen): ", m_pgen_input, 76);
	askStr ((char*)"Output file (Results): ", m_pgen_output, 76);
	askNum ((char*)"Line number", &m_pgen_line, 1, 999999999);

	if (askOkCancel ()) {
		IniWriteInt (INI_FILE, (char*)"Work", 0);
		IniWriteString (INI_FILE, (char*)"PgenInputFile", m_pgen_input);
		IniWriteString (INI_FILE, (char*)"PgenOutputFile", m_pgen_output);
		IniWriteInt (INI_FILE, (char*)"PgenLine", m_pgen_line);
		IniWriteInt (INI_FILE, (char*)"WorkDone", 0);
	}
}

/* Advanced/Priority dialog */

void advanced_priority ()
{
	unsigned long m_priority;

	m_priority = PRIORITY;

	outputLongLine ((char*)"Pick a priority between 1 and 10 where 1 is the lowest priority and 10 is the highest.\n");
	outputLongLine ((char*)"It is strongly recommended that you use the default priority of 1.  Your throughput will probably not improve by using a higher priority.  The only time you should raise the priority is when another process, such as a screen saver, is stealing CPU cycles from this program.\n");
	askNum ((char*)"Priority", &m_priority, 1, 10);

	if (askOkCancel ()) {
		PRIORITY = m_priority;
		IniWriteInt (INI_FILE, (char*)"Priority", PRIORITY);
	}
}

/* Options/CPU dialog */

void options_cpu () 
{ 
//	char buf[512]; 
 
//	getCpuDescription (buf, 0); 
//	printf ("CPU Information:\n%s\n", buf); 
	printf ("No CPU Information for now...\n"); 
 
	askOK (); 
} 

/* Options/Preferences dialog */

void options_preferences ()
{
	unsigned long m_iter, m_r_iter, m_disk_write_time;
	int	m_backup;

	m_iter = ITER_OUTPUT;
	m_r_iter = ITER_OUTPUT_RES;
	m_disk_write_time = DISK_WRITE_TIME;
	m_backup = TWO_BACKUP_FILES;

	askNum ((char*)"Iterations between screen outputs", &m_iter, 1, 999999999);
	askNum ((char*)"Iterations between results file outputs",
		&m_r_iter, 10000, 999999999);
	askNum ((char*)"Minutes between disk writes", &m_disk_write_time, 10, 999999);
	askYN ((char*)"Create Two Backup Files", &m_backup);

	if (askOkCancel ()) {
		ITER_OUTPUT = m_iter;
		ITER_OUTPUT_RES = m_r_iter;
		DISK_WRITE_TIME = m_disk_write_time;
		TWO_BACKUP_FILES = m_backup;
		IniWriteInt (INI_FILE, (char*)"OutputIterations", ITER_OUTPUT);
		IniWriteInt (INI_FILE, (char*)"ResultsFileIterations", ITER_OUTPUT_RES);
		IniWriteInt (INI_FILE, (char*)"DiskWriteTime", DISK_WRITE_TIME);
		IniWriteInt (INI_FILE, (char*)"TwoBackupFiles", TWO_BACKUP_FILES);
	}
}

/* Help/About */

void help_about ()
{
	printf ("Primality Testing of k*b^n+/-1 Program - GPU Version 3.8.7\n");
	printf ("Using George Woltman's 1/k method IBDWT for k's up to 19 bits\n"); 
	printf (" << Rational Base DWT for larger k's up to maximal double\n");
	printf (" << Rational base DWT and generic reduction for larger k's or base != 2\n");
        printf ("Using GPU code written by Shoichiro Yamada for IBDWT,\n << extended further to Rational Bases and Generic reduction.\n");
        printf ("By default, all tests on base two numbers use Gerbicz error checking.\nThis is the case for PRP Fermat and SPRP tests as in Prime95 or Mprime,\nbut also for the deterministic prime tests of Proth numbers.\nLLR tests on Riesel numbers are only done after a positive Fermat PRP result.\n");
	printf ("Written : Jan. 20011 - Oct. 2022 by Jean Penne\n");
	printf ("Email : jpenne@free.fr\n");
	askOK ();
}

/* Display the main menu */

void main_menu ()
{
	unsigned long choice;

mloop:	if (THREAD_KILL) return;
	printf ("\t     Main Menu\n");
loop:	printf ("\n");
	printf ("\t 1.  Test/Input Data\n");
	printf ("\t 2.  Test/Continue\n");
	printf ("\t 3.  Test/Exit\n");
	printf ("\n");
	printf ("\t 4.  Options/CPU\n");
	printf ("\t 5.  Options/Preferences\n");
	printf ("\t 6.  Advanced/Priority\n");
	printf ("\n");
	printf ("\t 7.  Help/About\n");
	printf ("Your choice: ");
	choice = get_number (0);
	if (choice <= 0 || choice >= 8) {
		printf ("\t     Invalid choice\n");
		goto loop;
	}

/* Display the main menu and switch off the users choice */

	printf ("\n");
	switch (choice) {

/* Test/Primenet dialog */

	case 1:
		test_inputdata ();
		break;

/* Test/Continue */

	case 2:
		linuxContinue ((char*)"Another llrCUDA is running.\n");
		askOK ();
		break;

/* Test/Exit */

	case 3:
		exit(0);  // JP 08/06/17
 //             return;

/* Options/CPU dialog */

	case 4:
		options_cpu ();
		break;

/* Options/Preferences dialog */

	case 5:
		options_preferences ();
		break;

/* Advanced/Priority dialog */

	case 6:
		advanced_priority ();
		break;

/* Help/About */

	case 7:
		help_about ();
		break;

	}
	goto mloop;
}
