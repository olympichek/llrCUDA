#include "hip/hip_runtime.h"
/**************************************************************
 *
 *	gwypnum.c
 *
 *  Modulo k*2^n+/-1 DWFFT multiplications and squarings prototype
 *  source code, GPU version ; must be used linked with cuda and cufft
 *  libraries (versions 8.0.44 work fine!).
 *  This code is built from the one included in Shoichiro Yamada's
 *  llrcuda.0.931 that was released the 07/11/2015 on
 *  www.mersenneforum.org, and, indeed, also from llrp version 3.8.1
 *  (the last released portable LLR version).
 * 
 *  Thanks to the nice work of Shoichiro, it was not difficult for me to
 *  extend his code to rational bases DWT, and also, to generic modular
 *  reduction ; that is what is done here!
 * 
 *  This code is fully C and C++ written, no Assembler code.
 *  Large numbers (at least 1 mega digits) benefit more from the GPU
 *  parallelism, but this program may also be used on smaller positive
 *  results for verification...
 * 
 *  Below is a bit of history of the llrp program :
 *  Nothing original here ; my goal was to have a code portable on any
 *  system having a C / C++ compiler,
 *  and, indeed a processor with a sufficiently powerful floating point
 *  unit!
 *  First : 14/09/2005 : uses George Woltman's 1/k IBDWT method and
 *  cyclic (c = -1), negacyclic (c = +1) real convolutions.
 *  Drawback in the negacyclic case : k must be small due to the 1/cos
 *  factor in the inverse DFFT.
 *  (this factor becomes large near the middle of the FFT array!)
 *  Updates:
 *  14/04/2008 : Full complex, half length convolution used when
 *  computing modulo k*2^n+1 .
 *  Nov. 2010 : zero-padded FFT implemented for k's up to 45 bits large.
 *  Dec. 2010 : generic modular reduction implemented for k's larger
 *  than 45 bits or for general form moduli.
 *  May. 2011 : This code must be linked with  Matteo Frigo and Steven
 *  G. Johnson's FFTW library.
 *  Thanks to this FFTW usage, a power of two FFT length is no more
 *  required.
 *  January 2018 : In this GPU version , CUFFT 8.0.44 is used for all
 *  Fourier transforms.
 *  Jean Penne  02/01/2018, E-mail : jpenne@free.fr
 *
 **************************************************************/

/* Include Files */

#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <assert.h>
#include "fftw3.h"
#if defined (__linux__) || defined (__FreeBSD__) || defined (__APPLE__)
#include <sys/time.h>
#define _timeb		timeb
#define _ftime		ftime
#else
#include <time.h>
#endif
#include "giants.h"
#include "gwdbldbl.h"
#include "gwypnum.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include "cuda_safecalls.h"


/* definitions */

#define BITS 16
#ifndef _MIPSBUILD_
#define MAXBITSPERDOUBLE (double)35
#define MAXKBITS (double)18
#else
#define MAXBITSPERDOUBLE (double)35
#define MAXKBITS (double)18
#endif

/* The maximum value k * mulbyconst that can be in a zero pad FFT.  Larger */
/* values must use generic modular reduction. */
/*
#if  defined (__linux__) || defined (__FreeBSD__) || defined (__APPLE__)
#define MAX_ZEROPAD_K	35184372088831.0	// 45-bit k's seem to be OK.
#else
#define MAX_ZEROPAD_K	68719476735.0		// 36-bit k's seem to be OK.
#endif
*/
#define MAX_ZEROPAD_K	5000000000.0           // Experimental

#define EB 10	// Extra bits of precision for generic reduction

#ifndef WIN32
#define LINE_FEED "\r"
#elif defined (_CONSOLE)
#define LINE_FEED "\r"
#else
#define LINE_FEED "\n"
#endif

/* Handy macros to improve readability */

#define log2(n)		(log((double)(n)) / log (2.0))
#define logb(n)		(log((double)(n)) / log ((double)(b)))
#define divide_rounding_up(a,b)		((a + (b) - 1) / (b))
#define divide_rounding_down(a,b)	((a) / (b))
#define round_up_to_multiple_of(a,b)	(divide_rounding_up (a, b) * (b))
#define round_down_to_multiple_of(a,b)	(divide_rounding_down (a, b) * (b))

/* MSVC6 has trouble with the pow function using integer arguments. */
/* For example, "(unsigned long) pow (5.0, 7.0)" returns 78124 instead */
/* of the correct 78125.  This macro, works around this trouble. */

#define intpow(b,n)	((long) floor (pow ((double)(b), (double)(n)) + 0.1))

/* global variables */

#include "gwtables.cu"

double gwyptimers[2*NBTIMERS] = {0.0};		/* Up to NBTIMERS separate timers */

// k, b, n, c as global variables :

extern giant gk;
extern double MAXBPD;
double kg;
unsigned long bg, ng;
signed long cg;

unsigned long NUM_B_PER_SMALL_WORD, GW_GEN_MOD_MAX;
unsigned long maxbitsinfftlen, maxbitsinfftword;

int     cufftonly = FALSE;
int	E_CHK = 0;   // JP 20/06/17
int	CUMULATIVE_TIMING = 0;
int     error_log = 0;   // JP 20/06/17
int     balerr = FALSE, setup = FALSE;
int     nb_malloc = 0, nb_free = 0, nb_cudamalloc = 0, nb_cudafree = 0;
double	MAXERR;
giant	gmodulus = NULL;
giant	grecip = NULL;
// giant	gtmp = NULL;
gwypnum	GWP_RANDOM = NULL;
gwypnum	modulus = NULL;
gwypnum	recip = NULL;
gwypnum	gwyptmp = NULL;
void	(*printfunction)(char*) = NULL;
void	(*screen_output)(char*) = NULL;
void	(*both_output)(char*) = NULL;
void    *dd_data_arg;

int	plus = 0;
int	compl2 = 0;
int	zp = 0;
int	generic = 0;
int	zcomplex = TRUE;
int	verbose = 0;

double	*cn=NULL, *sn=NULL, *cnp=NULL, *snp=NULL, *two_to_phi=NULL, *two_to_minusphi=NULL,
                *invlimit=NULL, *flimit=NULL,
		*hlimit=NULL, *limitbv=NULL;
double 	log2ofkbpown, high, low, highinv, lowinv, last, lastinv, addinvalue, wrapfactor;
double	BIGVAL, SMALLMULCONST = 1.0, MAXMULCONST = 1.0;
double	ttmp, avg_num_b_per_word;
double		*xin=NULL, *yin=NULL;
fftw_complex	*cxin=NULL, *cyin=NULL, *cxout=NULL, *cyout=NULL;
double	*cuda_xin=NULL;
double	*cuda_cxin=NULL;
double	*cuda_cxout=NULL;
double	*cuda_yin=NULL;
double	*cuda_cyin=NULL;
double	*cuda_cyout=NULL;
double *cuda_x=NULL;
double *cuda_y=NULL;
double *cuda_m=NULL;
double *cuda_r=NULL;
double *cuda_cm=NULL;
double *cuda_cr=NULL;
double *cuda_tmp=NULL;
double *cuda_tmp_g=NULL;
double *cuda_two_to_phi=NULL;
double *cuda_two_to_minusphi=NULL;
double *cuda_cnp=NULL;
double *cuda_snp=NULL;
double *g_limitbv=NULL,*g_invlimit=NULL,*g_carry=NULL,*g_hlimit=NULL;
float  *g_err=NULL;
float  *l_err=NULL;
double	BIGVAL2 = 6755399441055744.0; // 3*2^51
int     g_fftlen = 0;

fftw_plan		fwpx, fwpy, bwpx, bwpy;
hipfftHandle 		cuda_fwpx;
hipfftHandle 		cuda_bwpx;
unsigned long  bit_length, zerowordslow, zerowordshigh; 
int 	 *fftbase=NULL, addinindex, wrapindex;
int	FFTLEN = 0, debug = 0, tdebug = 0, MULBYCONST = 0, FFTINC = 0;
int s_FFTLEN = 0; //cuda
char	gwypbuf[256];

// Variables used for modular reduction in zero-padded mode

unsigned long temp = 0, rem, hwcount, lwcount, hwoffset, bits;
int		inc;
double	mult, invmult, shift, limit_high, limit_inverse_high, limit_high_bigval;
double	*scr=NULL, *scral=NULL;


/**************************************************************
 *
 *	Functions
 *
 **************************************************************/

/* rint is not ANSI compatible, so we need a definition for 
 * WIN32 and other platforms with rint.
 */

double
RINT(double x)
{
    return floor(x + 0.5);
}

// macro used only to compute BIGVAL :
#define RINTP(x) ((x)-BIGVAL)+BIGVAL 


// Allocation routine

gwypnum gwypalloc()
{
    return((gwypnum)malloc(FFTLEN*sizeof(double)));
}

void gwyp_clear_maxerr ()
{
    MAXERR = 0.0;
}

double gwyp_get_maxerr ()
{
    return (MAXERR);
}

// Utilities

void gwyptrace (int n) {
    printfunction = (verbose)? both_output : screen_output;
    sprintf (gwypbuf, "OK until number %d\n", n);
    if (printfunction != NULL)
        (*printfunction)(gwypbuf);
}

void gwypclearline (int size) {
    char buf[256];
    int i;
    for (i=0; i<256; i++)
        buf[i] = '\0';
    for (i=0; i<size; i++)
        buf[i] = ' ';
    buf[size-1] = '\r';
#if !defined(WIN32) || defined(_CONSOLE)
    printf("%s", buf);
#endif
}


void gwypgianttoula (giant a, unsigned long* b) {
    // Convert a positive giant in an unsigned long array
    long i;
    
    for (i = 0;i<(abs(a->sign)/2)+2;i++)
        b[i] = 0;
    if (a->sign <= 0)
        return;
    for (i = 0 ;i < a->sign;i += 2)
        *b++ = (unsigned long)a->n[i]|((unsigned long)a->n[i+1]<<16);
    if (a->sign & 1)        // odd number of 16 bits input words
        *(--b) = (unsigned long)a->n[a->sign-1]; // re-write last word.
}
    
int gwyptogiantnomodulo (
    gwypnum gg,
    giant   v)
{
    long        val;
    int         i, j, limit, bits, bitsout, carry;
    unsigned short *outptr;

/* If this is a zero padded FFT, then only convert a little more than half of the FFT data words. */
/* For a DWT, convert all the FFT data. */

    if (zp || generic)
        limit = FFTLEN / 2 + 4;
    else
        limit = FFTLEN;
    
/* GENERAL_MOD has some strange cases we must handle.  In particular the */
/* last fft word translated can be 2^bits and the next word could be -1, */
/* this must be translated into zero, zero. */

    if (generic) {
        long    val, prev_val;
        while (limit < FFTLEN) {
            val = (long)gg[limit];
            if (val == -1 || val == 0)
                break;
            limit++;
        }
        while (limit > 1) { /* Find top word */
            prev_val = (long)gg[limit-1];
            if (val != prev_val || val < -1 || val > 0)
                break;
            limit--;
        }
        limit++;
    }

/* If base is 2 we can simply copy the bits out of each FFT word */

    if (1/*bg == 2*/) {
        /* Collect bits until we have all of them */
        carry = 0;
        bitsout = 0;
        outptr = v->n;
        *outptr = 0;
        if (zp || generic)
            bits = ng / FFTLEN;
        for (i = 0; i < limit; i++) {
            val = (long) gg[i];
            if (!zp && !generic)
                bits = fftbase[i];
            val += carry;
            for (j = 0; j < bits; j++) {
                *outptr >>= 1;
                if (val & 1)
                    *outptr += 0x8000;
                val >>= 1;
                bitsout++;
                if (bitsout == 16) {
                    outptr++;
                    bitsout = 0;
                }
            }
		carry = val;
        }
/* Finish outputting the last word and any carry data */
        while (bitsout || (carry != -1 && carry != 0)) {
            *outptr >>= 1;
            if (carry & 1)
                *outptr += 0x8000;
            carry >>= 1;
            bitsout++;
            if (bitsout == 16) {
                outptr++;
                bitsout = 0;
            }
        }
/* Set the length */
        v->sign = (long) (outptr - v->n);
        while (v->sign && (v->n[v->sign-1] == 0)) v->sign--;
/* If carry is -1, the gwnum is negative.  Ugh.  Flip the bits and sign. */	
        if (carry == -1) {
            for (j = 0; j < v->sign; j++)
                v->n[j] = ~v->n[j];
            while (v->sign && (v->n[v->sign-1] == 0))
                v->sign--;
            iaddg (1, v);
            v->sign = -v->sign;
        }
    }

/* Otherwise (base is not 2) we must do a radix conversion */

    else {
        giantstruct     *array = NULL;
        unsigned short  *buf = NULL;
        giant           small_base = NULL;
        giant           large_base = NULL;
        unsigned long   i, gap, small_size,
                        last_small_size;
        array = (giantstruct *) malloc (limit *  
                sizeof (giantstruct));
        buf = (unsigned short *) malloc (limit * sizeof (unsigned short));
        small_base = popg ();
        large_base = popg ();
        for (i = 0; i < limit; i++) {
            long        val;
            val = (long)gg[i];
            array[i].n[0] = buf[i];
//	    setmaxsize(&array[i], limit);
            itog (val, &array[i]);
        }
/* Loop combining pairs into ever larger and larger numbers.  Do all but last combining pass. */
        gap = 1;
        while (gap + gap < limit) {
            small_size = gwfft_base (dd_data_arg, gap) - 1;
            if (gap == 1)
                itog (intpow (bg, small_size), small_base);
            else if (small_size == last_small_size * 2)
                squareg(small_base);
            else
                mulg(large_base, small_base);
            itog (bg, large_base);
            mulg (small_base, large_base);
            for (i = 0; i + gap < limit; i += gap + gap) {
                gtog (&array[i+gap], v);
                if (gwfft_base (dd_data_arg, i+gap) - gwfft_base (dd_data_arg, i) == small_size)
                    mulg (small_base, v);
                else
                    mulg (large_base, v);
                addg (v, &array[i]);
            }
            gap = gap << 1;
            last_small_size = small_size;
        }
/* Do the last combining pass, outputting result directly to v. */
        if (gwfft_base (dd_data_arg, gap) == small_size * 2 + 1)
            mulg (small_base, large_base);
        else
            squareg (large_base);
        gtog (&array[gap], v);
        mulg (large_base, v);
        addg (&array[0], v);
/* Clean up */
        free (array);
        free (buf);
        pushg (2);
    }
/* Since all gwnums are premultiplied by the inverse of k, we must now multiply by k to get the true result. */
    if (kg != 1.0) {
        giant	newg;
        newg = newgiant ((unsigned long) (bit_length / 16) + 64);
        gtog (gk, newg);
        mulg (v, newg);
        gtog (newg, v);
        gwypfree (newg);
    }
/* Return success */
    return (0);
}

/* Routines used to time code chunks */

void gwypclear_timers () {
    int	i;
    for (i = 0; i < 2*NBTIMERS; i++)
        gwyptimers[i] = 0.0;
}

void gwypclear_timer (
    int	i)
{
    gwyptimers[i] = 0.0;
}

void gwypstart_timer ( 
    int	i) 
{ 
    struct _timeb timeval; 
    if (i >= NBTIMERS)
        return;
    if (gwyptimers[i+NBTIMERS] != 0.0)			// to avoid double start...
        return;
/*  if (HIGH_RES_TIMER) { 
        gwyptimers[i] -= getHighResTimer (); 
    } else { */
        _ftime (&timeval); 
        gwyptimers[i] -= (double) timeval.time * 1000.0 + timeval.millitm; 
//  } 
    gwyptimers[i+NBTIMERS] = 1.0;			// to show that gwyptimers[i] is already started
} 
 
void gwypend_timer ( 
    int	i) 
{ 
    struct _timeb timeval; 
    if (i >= NBTIMERS)
        return;
    if (gwyptimers[i+NBTIMERS] == 0.0)			// to avoid double end...
        return;
/*  if (HIGH_RES_TIMER) { 
        gwyptimers[i] += getHighResTimer (); 
    } else { */
        _ftime (&timeval); 
        gwyptimers[i] += (double) timeval.time * 1000.0 + timeval.millitm; 
//  } 
    gwyptimers[i+NBTIMERS] = 0.0;			// to show that gwyptimers[i] is ended
} 
 
void gwypdivide_timer (
    int	i,
    int	j)
{
    gwyptimers[i] = gwyptimers[i] / j;
}

double gwyptimer_value ( 
    int	i) 
{ 
/*  if (HIGH_RES_TIMER) 
        return (gwyptimers[i] / getHighResTimerFrequency ()); 
    else */
        return (gwyptimers[i] / 1000.0); 
} 
 
void gwypprint_timer (
    int	i,
    int	flags)
{ 
    char	buf[40]; 
    double	t; 
 
    t = gwyptimer_value (i); 
    if (flags & TIMER_NL)
        if (t >= 1.0)  
            sprintf (buf, "%.3f sec."LINE_FEED"", t); 
        else 
            sprintf (buf, "%.3f ms."LINE_FEED"", t * 1000.0);
    else
        if (t >= 1.0)  
            sprintf (buf, "%.3f sec.", t); 
        else 
            sprintf (buf, "%.3f ms.", t * 1000.0);
    printfunction = screen_output;
    if (printfunction != NULL)
        (*printfunction)(buf);
    if (flags & TIMER_CLR)
        gwyptimers[i] = 0.0; 
    if ((flags & TIMER_OPT_CLR) && !CUMULATIVE_TIMING)
        gwyptimers[i] = 0.0; 
} 

void gwypwrite_timer (	// JP 23/11/07
    char* buf,
    int	i, 
    int	flags) 
{ 
    double	t; 
 
    t= gwyptimer_value (i); 
    if (flags & TIMER_NL)
        if (t >= 1.0)  
            sprintf (buf, "%.3f sec.\n", t); 
        else 
            sprintf (buf, "%.3f ms.\n", t * 1000.0);
    else
        if (t >= 1.0)  
            sprintf (buf, "%.3f sec.", t); 
        else 
            sprintf (buf, "%.3f ms.", t * 1000.0);
	if (flags & TIMER_CLR)
            gwyptimers[i] = 0.0; 
	if ((flags & TIMER_OPT_CLR) && !CUMULATIVE_TIMING)
            gwyptimers[i] = 0.0; 
} 

void
print(
    double *x,
    int  N
)
{
    int  zc;

    printfunction = (verbose)? both_output : screen_output;

    while (N >= 0)  {
        zc = 0;
        while (x[N] == 0) {
            zc++;
            N--;
        }
        if (zc) {
            sprintf(gwypbuf,"(%d zeroes) ",zc);
            if (printfunction != NULL)
                (*printfunction)(gwypbuf);
        }
        if (N >=0) {
            sprintf(gwypbuf, "%g  ",x[N]);
            if (printfunction != NULL)
                (*printfunction)(gwypbuf);
            N--;
        }
    }
    sprintf(gwypbuf, "||\n");
    if (printfunction != NULL)
        (*printfunction)(gwypbuf);
}

/* Routine that copy a gwypnum from */
/* source to dest while zeroing some lower FFT words */

__global__ void
cuda_gwypcopyzero_kernel (
    double *s,
    double *d,
    unsigned long n,
    unsigned long len)
{
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    register double zero = 0.0;
    register unsigned long 	i;
        
    i = threadID;
    if (i<n)
        d[i] = zero;
    else if (i<len)
        d[i] = s[i];
}

void gwypcopyzero (
    gwypnum	s,
    gwypnum	d,
    unsigned long n)
{
    register double zero = 0.0;
    register double *sptr = s + n;
    register double *dptr = d;
    register double *maxptr;

    if (tdebug)
        gwypstart_timer (4);
    maxptr = d + n;
    while (dptr < maxptr)
        *dptr++ = zero;
    maxptr = d + FFTLEN;
    while (dptr < maxptr)
        *dptr++ = *sptr++;
    if (tdebug)
        gwypend_timer (4);
}

/* Set a gwypnum to zero */

__global__ void
cuda_gwypzero_kernel (
    double *s,
    unsigned long len)
{
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    register double zero = 0.0;
    register unsigned long 	i;
    i = threadID;
    if (i<len)
        s[i] = zero;
}

void gwypzero (gwypnum s) {
    long j;

    for(j=0; j<FFTLEN; ++j)
        s[j] = 0;
    return;
}

/* Routine that zero some high words in a gwypnum */

__global__ void
cuda_gwypsetzero_kernel (
    double *s,
    unsigned long n,
    unsigned long len)
{
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    register double zero = 0.0;
    register unsigned long 	i;
        
    i = threadID+len-n;
    if (i<len)
        s[i] = zero;
}

void gwypsetzero (
    gwypnum s,
    unsigned long n)
{
    register double zero = 0.0;
    register double *sptr = s + FFTLEN - n;
    register double *maxptr = s + FFTLEN;

    if (tdebug)
        gwypstart_timer (4);
    while (sptr < maxptr)
        *sptr++ = zero;
    if (tdebug)
        gwypend_timer (4);
}

// User side large integers arithmetic operations

__global__ void
cuda_gwypcopy_kernel (
    double *s,
    double *d,
    int n)
{
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    register int i;
        
    i = threadID;
    if (i<n)
        d[i] = s[i];
}

void gwypcopy (
    gwypnum s,
    gwypnum d)
{
    int i;

    for (i=0; i<FFTLEN; i++)
        d[i] = s[i];
}

__global__ void
cuda_gwypaddquick_kernel (
    double *s,
    double *d,
    int n)
{
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    register int i;
        
    i = threadID;
    if (i<n)
        d[i] += s[i];
}

void gwypaddquick (
    gwypnum s,
    gwypnum d)
{
    int i;

    for (i=0; i<FFTLEN; i++)
        d[i] += s[i];
}

__global__ void
cuda_gwypsubquick_kernel (
    double *s,
    double *d,
    int n)
{
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    register int i;
        
    i = threadID;
    if (i<n)
        d[i] -= s[i];
}

void gwypsubquick (
    gwypnum s,
    gwypnum d)
{
    int i;

    for (i=0; i<FFTLEN; i++)
        d[i] -= s[i];
}

// These functions do the relevant dyadic multiplications or squarings on Fourier transformed data

__global__ void
cuda_mul_complex_kernel(
    double *a,
    double *b,
    int n
)
{
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    register int k;
    register double Reb;

    k=threadID;
    if (k<n) {
        k=k*2;
        Reb = a[k]*b[k]-a[k+1]*b[k+1];
        b[k+1] = a[k+1]*b[k]+a[k]*b[k+1];
        b[k] = Reb;
    }
}

void
_mul_complex(
    fftw_complex *a,
    fftw_complex *b,
	int n
)
{
    register int k;
    register double Reb;

    for (k=0; k<n; k++) {
        Reb = a[k][0]*b[k][0]-a[k][1]*b[k][1];
        b[k][1] = a[k][1]*b[k][0]+a[k][0]*b[k][1];
        b[k][0] = Reb;
    }
}

__global__ void
cuda_square_complex_kernel(
    double *b,
    int n
)
{
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    register int k;
    register double Reb;
    k=threadID;
    if(k<n) {
        k=k*2;
        Reb = b[k]*b[k]-b[k+1]*b[k+1];
        b[k+1] = 2*b[k+1]*b[k];
        b[k] = Reb;
    }
}

void
cuda_square_complex(
    fftw_complex *b,
    int n
)
{
    register int k;
    register double Reb;

    for (k=0; k<n; k++) {
        Reb = b[k][0]*b[k][0]-b[k][1]*b[k][1];
        b[k][1] = 2*b[k][1]*b[k][0];
        b[k][0] = Reb;
    }

}



void
_square_complex(
    fftw_complex *b,
    int n
)
{
    register int k;
    register double Reb;

    for (k=0; k<n; k++) {
        Reb = b[k][0]*b[k][0]-b[k][1]*b[k][1];
        b[k][1] = 2*b[k][1]*b[k][0];
        b[k][0] = Reb;
    }
}

// These functions do the general multiplication or squaring of large integers, using DFFT

__global__ void
cuda_cnp_m_snp_kernel(
    double *cxin,
    double *cnp,
    double *snp,
    int n
)
{
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    register int k,j;
    register double ReX;
    j=threadID;
    if(j<n) {
        k=j*2;
        ReX = cnp[j]*cxin[k]-snp[j]*cxin[k+1];
        cxin[k+1] = cnp[j]*cxin[k+1]+snp[j]*cxin[k];
        cxin[k] = ReX;
    }
}

__global__ void
cuda_cnp_p_snp_kernel(
    double *cxin,
    double *cnp,
    double *snp,
    int n
)
{
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    register int k,j;
    register double ReX;
    j=threadID;
    if(j<n) {
        k=j*2;
        ReX = cnp[j]*cxin[k]+snp[j]*cxin[k+1];
        cxin[k+1] = cnp[j]*cxin[k+1]-snp[j]*cxin[k];
        cxin[k] = ReX;
    }
}

void cuda_fftwsquare_g (
    int size)
{
    if (compl2) {  // Full complex, half size DWFFT
        cuda_cnp_m_snp_kernel<<<(size/2+127)/128,128>>>(cuda_cxin,cuda_cnp,cuda_snp, size/2);
        if (tdebug)
            gwypstart_timer(6);
        cufftSafeCall(hipfftExecZ2Z(cuda_fwpx,(hipfftDoubleComplex *)cuda_cxin,(hipfftDoubleComplex *)cuda_cxout,HIPFFT_FORWARD));
        if (tdebug)
            gwypend_timer(6);
        cuda_square_complex_kernel<<<(size/2+127)/128,128>>>(cuda_cxout, size/2);
        if (tdebug)
            gwypstart_timer(6);
        cufftSafeCall(hipfftExecZ2Z(cuda_fwpx,(hipfftDoubleComplex *)cuda_cxout,(hipfftDoubleComplex *)cuda_cxin,HIPFFT_BACKWARD));
        if (tdebug)
            gwypend_timer(6);
        cuda_cnp_p_snp_kernel<<<(size/2+127)/128,128>>>(cuda_cxin,cuda_cnp,cuda_snp, size/2);
    }
    else  { // Real to complex, full size DWFFT
        if (tdebug)
            gwypstart_timer(6);
        cufftSafeCall(hipfftExecD2Z(cuda_fwpx,(hipfftDoubleReal *)cuda_xin,(hipfftDoubleComplex *)cuda_cxout));
        if (tdebug)
            gwypend_timer(6);
        cuda_square_complex_kernel<<<(size/2+1+127)/128,128>>>(cuda_cxout, size/2+1);
        if (tdebug)
            gwypstart_timer(6);
        cufftSafeCall(hipfftExecZ2D(cuda_bwpx,(hipfftDoubleComplex *)cuda_cxout,(hipfftDoubleReal *)cuda_xin));
        if (tdebug)
            gwypend_timer(6);
    }	
}

void fftwsquare_g (
    int size)
{
    register int j;
    register double ReX;

    if (compl2) {// Full complex, half size DWFFT
        // Multiply cxin by exp(i*j*pi/size) to prepare a right-angle convolution
        for (j=0; j<size/2; j++) {
            ReX = cnp[j]*cxin[j][0]-snp[j]*cxin[j][1];
            cxin[j][1] = cnp[j]*cxin[j][1]+snp[j]*cxin[j][0];
            cxin[j][0] = ReX;
        }

    }
    //fftw_execute (fwpx);	// Execute the relevant forward DFFT
    if (compl2) {
        if (tdebug)
            gwypstart_timer(6);
        cutilSafeCall(hipMemcpy(cuda_cxin,cxin,sizeof(double)*size,hipMemcpyHostToDevice));
        cufftSafeCall(hipfftExecZ2Z(cuda_fwpx,(hipfftDoubleComplex *)cuda_cxin,(hipfftDoubleComplex *)cuda_cxout,HIPFFT_FORWARD));
        cutilSafeCall(hipMemcpy(cxout,cuda_cxout,sizeof(double)*size,hipMemcpyDeviceToHost));
        if (tdebug)
            gwypend_timer(6);
    }
    else  {
        if (tdebug)
            gwypstart_timer(6);
        cutilSafeCall(hipMemcpy(cuda_xin,xin,sizeof(double)*size,hipMemcpyHostToDevice));
        cufftSafeCall(hipfftExecD2Z(cuda_fwpx,(hipfftDoubleReal *)cuda_xin,(hipfftDoubleComplex *)cuda_cxout));
        cutilSafeCall(hipMemcpy(cxout,cuda_cxout,sizeof(double)*2*(size/2+1),hipMemcpyDeviceToHost));
        if (tdebug)
            gwypend_timer(6);
    }
    if (compl2)// Compute the relevant Dyadic squaring
        _square_complex (cxout, size/2);
    else
        _square_complex(cxout, size/2+1);

    //fftw_execute (bwpx);// Execute the relevant backward DFFT
    if (compl2)  {
        if (tdebug)
            gwypstart_timer(6);
        cutilSafeCall(hipMemcpy(cuda_cxout,cxout,sizeof(double)*size,hipMemcpyHostToDevice));
        cufftSafeCall(hipfftExecZ2Z(cuda_fwpx,(hipfftDoubleComplex *)cuda_cxout,(hipfftDoubleComplex *)cuda_cxin,HIPFFT_BACKWARD));
        cutilSafeCall(hipMemcpy(cxin,cuda_cxin,sizeof(double)*size,hipMemcpyDeviceToHost));
        if (tdebug)
            gwypend_timer(6);
    }
    else    {
        if (tdebug)
            gwypstart_timer(6);
        cutilSafeCall(hipMemcpy(cuda_cxout,cxout,sizeof(double)*2*(size/2+1),hipMemcpyHostToDevice));
        cufftSafeCall(hipfftExecZ2D(cuda_bwpx,(hipfftDoubleComplex *)cuda_cxout,(hipfftDoubleReal *)cuda_xin));
        cutilSafeCall(hipMemcpy(xin,cuda_xin,sizeof(double)*size,hipMemcpyDeviceToHost));
        if (tdebug)
            gwypend_timer(6);
    }
    if (compl2) {   // Full complex, half size DWFFT
            // Multiply cxin by exp(-i*j*pi/size) to complete the right-angle convolution
        for (j=0; j<size/2; ++j)    {
            ReX = cnp[j]*cxin[j][0]+snp[j]*cxin[j][1];
            cxin[j][1] = cnp[j]*cxin[j][1]-snp[j]*cxin[j][0];
            cxin[j][0] = ReX;
        }
    }

}

void cuda_fftwmul_g (
    int size)
{
    if (compl2) {   // Full complex, half size DWFFT
        if (tdebug)
            gwypstart_timer(6);
        cuda_cnp_m_snp_kernel<<<(size/2+127)/128,128>>>(cuda_cxin,cuda_cnp,cuda_snp, size/2);
        cufftSafeCall(hipfftExecZ2Z(cuda_fwpx,(hipfftDoubleComplex *)cuda_cxin,(hipfftDoubleComplex *)cuda_cxout,HIPFFT_FORWARD));
        cuda_cnp_m_snp_kernel<<<(size/2+127)/128,128>>>(cuda_cyin,cuda_cnp,cuda_snp, size/2);
        cufftSafeCall(hipfftExecZ2Z(cuda_fwpx,(hipfftDoubleComplex *)cuda_cyin,(hipfftDoubleComplex *)cuda_cyout,HIPFFT_FORWARD));
        if (tdebug)
            gwypend_timer(6);
        cuda_mul_complex_kernel<<<(size/2+127)/128,128>>>(cuda_cxout, cuda_cyout, size/2);
        if (tdebug)
            gwypstart_timer(6);
        cufftSafeCall(hipfftExecZ2Z(cuda_fwpx,(hipfftDoubleComplex *)cuda_cyout,(hipfftDoubleComplex *)cuda_cyin,HIPFFT_BACKWARD));
        cuda_cnp_p_snp_kernel<<<(size/2+127)/128,128>>>(cuda_cyin,cuda_cnp,cuda_snp, size/2);
        if (tdebug)
            gwypend_timer(6);
    }
    else    {
        if (tdebug)
            gwypstart_timer(6);
        cufftSafeCall(hipfftExecD2Z(cuda_fwpx,(hipfftDoubleReal *)cuda_xin,(hipfftDoubleComplex *)cuda_cxout));
        cufftSafeCall(hipfftExecD2Z(cuda_fwpx,(hipfftDoubleReal *)cuda_yin,(hipfftDoubleComplex *)cuda_cyout));
        if (tdebug)
            gwypend_timer(6);
        cuda_mul_complex_kernel<<<(size/2+1+127)/128,128>>>(cuda_cxout, cuda_cyout, size/2+1);
        if (tdebug)
            gwypstart_timer(6);
        cufftSafeCall(hipfftExecZ2D(cuda_bwpx,(hipfftDoubleComplex *)cuda_cyout,(hipfftDoubleReal *)cuda_yin));
        if (tdebug)
            gwypend_timer(6);
    }	
}

void cuda_fftwmulbym_g (
    int size)
{
    if (compl2) {   // Full complex, half size DWFFT
        if (tdebug)
            gwypstart_timer(6);
        cuda_cnp_m_snp_kernel<<<(size/2+127)/128,128>>>(cuda_cyin,cuda_cnp,cuda_snp, size/2);
        cufftSafeCall(hipfftExecZ2Z(cuda_fwpx,(hipfftDoubleComplex *)cuda_cyin,(hipfftDoubleComplex *)cuda_cyout,HIPFFT_FORWARD));
        if (tdebug)
            gwypend_timer(6);
        cuda_mul_complex_kernel<<<(size/2+127)/128,128>>>(cuda_cm, cuda_cyout, size/2);
        if (tdebug)
            gwypstart_timer(6);
        cufftSafeCall(hipfftExecZ2Z(cuda_fwpx,(hipfftDoubleComplex *)cuda_cyout,(hipfftDoubleComplex *)cuda_cyin,HIPFFT_BACKWARD));
        cuda_cnp_p_snp_kernel<<<(size/2+127)/128,128>>>(cuda_cyin,cuda_cnp,cuda_snp, size/2);
        if (tdebug)
            gwypend_timer(6);
    }
    else    {
        if (tdebug)
            gwypstart_timer(6);
        cufftSafeCall(hipfftExecD2Z(cuda_fwpx,(hipfftDoubleReal *)cuda_yin,(hipfftDoubleComplex *)cuda_cyout));
        if (tdebug)
            gwypend_timer(6);
        cuda_mul_complex_kernel<<<(size/2+1+127)/128,128>>>(cuda_cm, cuda_cyout, size/2+1);
        if (tdebug)
            gwypstart_timer(6);
        cufftSafeCall(hipfftExecZ2D(cuda_bwpx,(hipfftDoubleComplex *)cuda_cyout,(hipfftDoubleReal *)cuda_yin));
        if (tdebug)
            gwypend_timer(6);
    }	
}

void cuda_fftwmulbyr_g (
    int size)
{
    if (compl2) {   // Full complex, half size DWFFT
        if (tdebug)
            gwypstart_timer(6);
        cuda_cnp_m_snp_kernel<<<(size/2+127)/128,128>>>(cuda_cyin,cuda_cnp,cuda_snp, size/2);
        cufftSafeCall(hipfftExecZ2Z(cuda_fwpx,(hipfftDoubleComplex *)cuda_cyin,(hipfftDoubleComplex *)cuda_cyout,HIPFFT_FORWARD));
        if (tdebug)
            gwypend_timer(6);
        cuda_mul_complex_kernel<<<(size/2+127)/128,128>>>(cuda_cr, cuda_cyout, size/2);
        if (tdebug)
            gwypstart_timer(6);
        cufftSafeCall(hipfftExecZ2Z(cuda_fwpx,(hipfftDoubleComplex *)cuda_cyout,(hipfftDoubleComplex *)cuda_cyin,HIPFFT_BACKWARD));
        cuda_cnp_p_snp_kernel<<<(size/2+127)/128,128>>>(cuda_cyin,cuda_cnp,cuda_snp, size/2);
        if (tdebug)
            gwypend_timer(6);
	}
    else    {
        if (tdebug)
            gwypstart_timer(6);
        cufftSafeCall(hipfftExecD2Z(cuda_fwpx,(hipfftDoubleReal *)cuda_yin,(hipfftDoubleComplex *)cuda_cyout));
        if (tdebug)
            gwypend_timer(6);
        cuda_mul_complex_kernel<<<(size/2+1+127)/128,128>>>(cuda_cr, cuda_cyout, size/2+1);
        if (tdebug)
            gwypstart_timer(6);
        cufftSafeCall(hipfftExecZ2D(cuda_bwpx,(hipfftDoubleComplex *)cuda_cyout,(hipfftDoubleReal *)cuda_yin));
        if (tdebug)
            gwypend_timer(6);
    }	
}

void fftwmul_g (int size) {
    register int j;
    register double ReX, ReY;
    if (compl2) {  // Full complex, half size DWFFT
        // Multiply cxin and cyin by exp(i*j*pi/size) to prepare a right-angle convolution
        for (j=0; j<size/2; j++) {
            ReX = cnp[j]*cxin[j][0]-snp[j]*cxin[j][1];
            cxin[j][1] = cnp[j]*cxin[j][1]+snp[j]*cxin[j][0];
            cxin[j][0] = ReX;
            ReY = cnp[j]*cyin[j][0]-snp[j]*cyin[j][1];
            cyin[j][1] = cnp[j]*cyin[j][1]+snp[j]*cyin[j][0];
            cyin[j][0] = ReY;
        }
    }
    if (compl2) {
	//fftw_execute (fwpx);	// Execute the two relevant forward DFFTs
        if (tdebug)
            gwypstart_timer(6);
        cutilSafeCall(hipMemcpy(cuda_cxin,cxin,sizeof(double)*size,hipMemcpyHostToDevice));
	cufftSafeCall(hipfftExecZ2Z(cuda_fwpx,(hipfftDoubleComplex *)cuda_cxin,(hipfftDoubleComplex *)cuda_cxout,HIPFFT_FORWARD));
	cutilSafeCall(hipMemcpy(cxout,cuda_cxout,sizeof(double)*2*(size/2+1),hipMemcpyDeviceToHost));
	//fftw_execute (fwpy);
	cutilSafeCall(hipMemcpy(cuda_cyin,cyin,sizeof(double)*size,hipMemcpyHostToDevice));
	cufftSafeCall(hipfftExecZ2Z(cuda_fwpx,(hipfftDoubleComplex *)cuda_cyin,(hipfftDoubleComplex *)cuda_cyout,HIPFFT_FORWARD));
        cutilSafeCall(hipMemcpy(cyout,cuda_cyout,sizeof(double)*2*(size/2+1),hipMemcpyDeviceToHost));
        if (tdebug)
            gwypend_timer(6);
    }
    else {
	//fftw_execute (fwpx);	// Execute the two relevant forward DFFTs
        if (tdebug)
            gwypstart_timer(6);
        cutilSafeCall(hipMemcpy(cuda_xin,xin,sizeof(double)*size,hipMemcpyHostToDevice));
	cufftSafeCall(hipfftExecD2Z(cuda_fwpx,(hipfftDoubleReal *)cuda_xin,(hipfftDoubleComplex *)cuda_cxout));	cutilSafeCall(hipMemcpy(cxout,cuda_cxout,sizeof(double)*2*(size/2+1),hipMemcpyDeviceToHost));	cutilSafeCall(hipMemcpy(cuda_yin,yin,sizeof(double)*size,hipMemcpyHostToDevice));
	cufftSafeCall(hipfftExecD2Z(cuda_fwpx,(hipfftDoubleReal *)cuda_yin,(hipfftDoubleComplex *)cuda_cyout));	cutilSafeCall(hipMemcpy(cyout,cuda_cyout,sizeof(double)*2*(size/2+1),hipMemcpyDeviceToHost));
        if (tdebug)
            gwypend_timer(6);
    }

    if (compl2)	// Compute the relevant Dyadic product
        _mul_complex (cxout, cyout, size/2);
    else
        _mul_complex(cxout, cyout, size/2+1);
    if (compl2) {
        if (tdebug)
            gwypstart_timer(6);
        cutilSafeCall(hipMemcpy(cuda_cyout,cyout,sizeof(double)*size,hipMemcpyHostToDevice));
        cufftSafeCall(hipfftExecZ2Z(cuda_fwpx,(hipfftDoubleComplex *)cuda_cyout,(hipfftDoubleComplex *)cuda_cyin,HIPFFT_BACKWARD));
        cutilSafeCall(hipMemcpy(cyin,cuda_cyin,sizeof(double)*size,hipMemcpyDeviceToHost));
        if (tdebug)
            gwypend_timer(6);
    }
    else {
	//fftw_execute (bwpy);	// Execute the relevant backward DFFT
        if (tdebug)
            gwypstart_timer(6);
        cutilSafeCall(hipMemcpy(cuda_cyout,cyout,sizeof(double)*2*(size/2+1),hipMemcpyHostToDevice));
	cufftSafeCall(hipfftExecZ2D(cuda_bwpx,(hipfftDoubleComplex *)cuda_cyout,(hipfftDoubleReal *)cuda_yin));	cutilSafeCall(hipMemcpy(yin,cuda_yin,sizeof(double)*size,hipMemcpyDeviceToHost));
        if (tdebug)
            gwypend_timer(6);
    }
    if (compl2) {  // Full complex, half size DWFFT
        // Multiply cyin by exp(-i*j*pi/size) to complete the right-angle convolution
        for (j=0; j<size/2; ++j)    {
            ReY = cnp[j]*cyin[j][0]+snp[j]*cyin[j][1];
            cyin[j][1] = cnp[j]*cyin[j][1]-snp[j]*cyin[j][0];
            cyin[j][0] = ReY;
        }
    }
}

#define STRIDE_DIM 256
/**************************************************************
 *
 *      Functions
 *
 **************************************************************/
#define BLOCK_DIM 16

// This kernel is optimized to ensure all global reads and writes are coalesced,
// and to avoid bank conflicts in shared memory.  This kernel is up to 11x faster
// than the naive kernel below.  Note that the shared memory array is sized to 
// (BLOCK_DIM+1)*BLOCK_DIM.  This pads each row of the 2D block in shared memory 
// so that bank conflicts do not occur when threads address the array column-wise.

__global__ void transpose(double *odata, double *idata, int width, int height)
{
    __shared__ double block[BLOCK_DIM][BLOCK_DIM+1];

    // read the matrix tile into shared memory
    unsigned int xIndex = blockIdx.x * BLOCK_DIM + threadIdx.x;
    unsigned int yIndex = blockIdx.y * BLOCK_DIM + threadIdx.y;
    if((xIndex < width) && (yIndex < height))   {
        unsigned int index_in = yIndex * width + xIndex;
        block[threadIdx.y][threadIdx.x] = idata[index_in];
    }
    __syncthreads();
    // write the transposed matrix tile to global memory
    xIndex = blockIdx.y * BLOCK_DIM + threadIdx.x;
    yIndex = blockIdx.x * BLOCK_DIM + threadIdx.y;
    if((xIndex < height) && (yIndex < width))   {
        unsigned int index_out = yIndex * height + xIndex;
        odata[index_out] = block[threadIdx.x][threadIdx.y];
    }
}

__global__ void mul_const_transpose(double *odata, double *idata, double c, int width, int height)
{
    __shared__ double block[BLOCK_DIM][BLOCK_DIM+1];
    // read the matrix tile into shared memory
    unsigned int xIndex = blockIdx.x * BLOCK_DIM + threadIdx.x;
    unsigned int yIndex = blockIdx.y * BLOCK_DIM + threadIdx.y;
    if((xIndex < width) && (yIndex < height))   {
        unsigned int index_in = yIndex * width + xIndex;
        block[threadIdx.y][threadIdx.x] = idata[index_in]*c;
    }
    __syncthreads();
    // write the transposed matrix tile to global memory
    xIndex = blockIdx.y * BLOCK_DIM + threadIdx.x;
    yIndex = blockIdx.x * BLOCK_DIM + threadIdx.y;
    if((xIndex < height) && (yIndex < width))   {
        unsigned int index_out = yIndex * height + xIndex;
        odata[index_out] = block[threadIdx.x][threadIdx.y];
    }
}

__global__ void mul_0_transpose(double *odata, double *idata, double *mul, int width, int height)
{
    __shared__ double block[BLOCK_DIM][BLOCK_DIM+1];
    // read the matrix tile into shared memory
    unsigned int xIndex = blockIdx.x * BLOCK_DIM + threadIdx.x;
    unsigned int yIndex = blockIdx.y * BLOCK_DIM + threadIdx.y;
    if((xIndex < width) && (yIndex < height))   {
        unsigned int index_in = yIndex * width + xIndex;
        block[threadIdx.y][threadIdx.x] = idata[index_in]*mul[index_in];
    }
    __syncthreads();
    // write the transposed matrix tile to global memory
    xIndex = blockIdx.y * BLOCK_DIM + threadIdx.x;
    yIndex = blockIdx.x * BLOCK_DIM + threadIdx.y;
    if((xIndex < height) && (yIndex < width))   {
        unsigned int index_out = yIndex * height + xIndex;
        odata[index_out] = block[threadIdx.x][threadIdx.y];
    }
}

__global__ void mul_1_transpose(double *odata, double *idata, double *mul, int width, int height)
{
    __shared__ double block[BLOCK_DIM][BLOCK_DIM+1];
    // read the matrix tile into shared memory
    unsigned int xIndex = blockIdx.x * BLOCK_DIM + threadIdx.x;
    unsigned int yIndex = blockIdx.y * BLOCK_DIM + threadIdx.y;
    if((xIndex < width) && (yIndex < height))   {
        unsigned int index_in = yIndex * width + xIndex;
        block[threadIdx.y][threadIdx.x] = idata[index_in];
    }
    __syncthreads();
    // write the transposed matrix tile to global memory
    xIndex = blockIdx.y * BLOCK_DIM + threadIdx.x;
    yIndex = blockIdx.x * BLOCK_DIM + threadIdx.y;
    if((xIndex < height) && (yIndex < width))   {
        unsigned int index_out = yIndex * height + xIndex;
        odata[index_out] = block[threadIdx.x][threadIdx.y]*mul[index_out];
    }
}

//#define IDX(i) ((((i) >> 14) << 14) + (((i) & (128*128-1)) >> 7)  + (((i) & 127) << 7))
#define IDX(i) ((((i) >> 16) << 16) + (((i) & (256*256-1)) >> 8)  + (((i) & 255) << 8))
//#define IDX(i) ((((i) >> 18) << 18) + (((i) & (512*512-1)) >> 9)  + (((i) & 511) << 9))

// Functions that do the normalization of the result of a multiplication or squaring

__global__ void cuda_inormalize_kernel(
    // Used for irrational bases DWFFT
    double  *x,
    int     N,
    int     error_log,
    int     noadd,
    int     nomul,
    double  *g_limitbv,
    double  *g_limitbv2,
    double  *g_invlimit,
    int     STRIDE,
    double  bv,
    double  bv2,
    double  *g_carry,
    float   *g_err,
    int     wrapindex,
    double  wrapfactor,
    int     MULBYCONST,
    double  SMALLMULCONST,
    int     addinindex,
    double  addinvalue,
    int     cg      // 25/03/21
)
{
    register int    j;
    register double xx,zz,tx;
    register double carry = 0.0,err = 0.0,maxerr = 0.0;
    const int 	threadID = blockIdx.x * blockDim.x + threadIdx.x;
    if(addinindex==0)   {
        if( threadID==0)
            if (!noadd) 
                x[IDX(addinindex)] += addinvalue;   // Add the optional small constant
    }
    else    {
        if( threadID*STRIDE < N && threadID*STRIDE+STRIDE >= N)
            if (!noadd) 
                x[IDX(addinindex)] += addinvalue;   // Add the optional small constant
    }
    carry = 0;
    maxerr = 0;
    for (j = threadID*STRIDE; j < threadID*STRIDE+STRIDE && j < N;++j)  {
        tx = x[IDX(j)];
        if(MULBYCONST && !nomul)
            tx *= SMALLMULCONST;
        // Optionaly multiply by a small constant
        xx = (tx + bv) - bv2;
        // Round to the nearest integer
        if(error_log) {
        // Compute the rounding error if required
            if(xx<0)
                err = fabs(-xx + tx);
            else
                err = fabs(xx  - tx);
            if(err > maxerr)
                maxerr = err;
        }
        xx += carry;
        zz = (xx+g_limitbv[IDX(j)])-g_limitbv2[IDX(j)];
        carry = zz*g_invlimit[IDX(j)];
        // Compute the carry on next word
        x[IDX(j)] = xx - zz;
        // And the balanced remainder in current one
    }
    g_carry[threadID]=carry;
    if(g_err[threadID] < maxerr)
    g_err[threadID] = maxerr;
}

__global__ void cuda_inormalize2_kernel(
    // Used for irrational bases DWFFT
    double  *x,
    int     N,
    int     error_log,
    int     noadd,
    int     nomul,
    double  *g_limitbv,
    double  *g_limitbv2,
    double  *g_invlimit,
    int     STRIDE,
    double  bv,
    double  bv2,
    double  *g_carry,
    float   *g_err,
    int     wrapindex,
    double  wrapfactor,
    int     MULBYCONST,
    double  SMALLMULCONST,
    int     addinindex,
    double  addinvalue,
    int     cg      // 24/03/21
)
{
    register int    j;
    register double tx,xx,zz;
    register double carry,carry2;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

    if(threadID*STRIDE < N && threadID*STRIDE+STRIDE >= N)  {
        carry = g_carry[threadID];
        carry2 = 0;
        if (carry)  {
            j = 0;
            if (wrapindex)
                carry2 = carry*wrapfactor;
//            if (plus) 
//                carry = -carry;
            carry *= -cg;    // accept abs(c)!=1 here! 24/03/21
            while (carry||carry2)   {       
                if (wrapindex && !carry)
                // Skip already normalized words
                    j = wrapindex;
                tx = x[IDX(j)];
                xx = tx + carry;
                if (wrapindex && j==wrapindex) {
                    xx += carry2;
                    carry2 = 0.0;
                }
                zz = (xx+g_limitbv[IDX(j)])-g_limitbv2[IDX(j)];
                carry = zz*g_invlimit[IDX(j)];
                // Compute the carry on next word
                x[IDX(j)] = xx - zz;
                // And the balanced remainder in current word
                if (++j == N)   {
                    j = 0;
                    if (wrapindex)
                        carry2 = carry*wrapfactor;
//                    if (plus)		
//                        carry = -carry;
                    carry *= -cg;    // accept abs(c)!=1 here! 24/03/21
                }
            }
        }
    }
    else    {
        carry = g_carry[threadID];
        carry2 = 0;
        j = threadID*STRIDE+STRIDE;
        while (carry)   {
            tx = x[IDX(j)];
            xx = tx + carry;
            zz = (xx+g_limitbv[IDX(j)])-g_limitbv2[IDX(j)];
            carry = zz*g_invlimit[IDX(j)];          // Compute the carry on next word
            x[IDX(j)] = xx - zz;                      // And the balanced remainder in current word
            j++;
        }
    }
}

double
inormalize(  // Used for irrational bases DWFFT
    double *x,
    int     N,
    int	    error_log,
    int	    noadd,
    int	    nomul
)
{
    register int j;
    register double *px = x, xx, zz, bv = BIGVAL2;
        // JP 08/07/17
    register double  carry = 0.0, carry2 = 0.0, err, maxerr = 0.0;

    if (!noadd)
        x[addinindex] += addinvalue;
        // Add the optional small constant
    for (j=0; j<N; ++j) {
        if (MULBYCONST && !nomul)
            *px *= SMALLMULCONST;
        // Optionaly multiply by a small constant
        xx = (*px + bv) - bv;
        // Round to the nearest integer
        if (error_log ) {
        // Compute the rounding error if required
            if (xx<0)
                err = fabs(-xx + *px);  
            else 
                err = fabs(xx  - *px);
            if (err > maxerr) 
                maxerr = err;
        }
        xx += carry;
        zz = (xx+limitbv[j])-limitbv[j];
        carry = zz*invlimit[j];
        // Compute the carry on next word
        *(px++) = xx - zz;
        // And the balanced remainder in current one
    }
    if (carry)  {
        j = 0;
        px = x;
        if (wrapindex)
            carry2 = carry*wrapfactor;
//        if (plus)
//            carry = -carry;
        carry *= -cg;    // accept abs(c)!=1 here! 24/03/21
        while (carry||carry2)   {
            if (wrapindex && !carry) {
                // Skip already normalized words
                j = wrapindex;
                px = x + wrapindex;
            }
            xx = *px + carry;
            if (wrapindex && j==wrapindex) {
                xx += carry2;
                carry2 = 0.0;
            }

            zz = (xx+limitbv[j])-limitbv[j];
            carry = zz*invlimit[j];
            // Compute the carry on next word
            *(px++) = xx - zz;
            // And the balanced remainder in current word
            if (++j == N)   {
                j = 0;
                px = x;
                if (wrapindex)
                    carry2 = carry*wrapfactor;
//                if (plus)
//                    carry = -carry;
                carry *= -cg;    // accept abs(c)!=1 here! 24/03/21
            }
        }
    }
    return(maxerr);
}

//      Functions that do the normalization of the result of a multiplication or squaring

__global__ void cuda_rnormalize_kernel(
    // Used for rational bases DWFFT
    double  *x,
    int     N,
    int     error_log,
    int     noadd,
    int     nomul,
    double  limitbv,
    double  limitbv2,
    double  invlimit,
    int     STRIDE,
    double  bv,
    double  bv2,
    double  *g_carry,
    float   *g_err,
    int     MULBYCONST,
    double  SMALLMULCONST,
    int     addinindex,
    double  addinvalue,
    int     cg      // 25/03/21
)
{
    register int    j;
    register double xx, zz, tx;
    register double carry = 0.0, err, maxerr = 0.0;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    if(addinindex==0)   {
        if(threadID==0)
            if (!noadd) 
                x[IDX(addinindex)] += addinvalue;   // Add the optional small constant
    }
    else    {
        if( threadID*STRIDE < N && threadID*STRIDE+STRIDE >= N)
            if (!noadd) 
                x[IDX(addinindex)] += addinvalue;   // Add the optional small constant
    }
    carry = 0;
    maxerr = 0;
    for (j = threadID*STRIDE; j < threadID*STRIDE+STRIDE && j < N;++j)  {
        tx = x[IDX(j)];
        if(MULBYCONST && !nomul)
            tx *= SMALLMULCONST;
        // Optionaly multiply by a small constant
        xx = (tx + bv) - bv2;
        // Round to the nearest integer
        if(error_log) {
        // Compute the rounding error if required
            if(xx<0)
                err = fabs(-xx + tx);
            else
                err = fabs(xx  - tx);
            if(err > maxerr)
                maxerr = err;
        }
        xx += carry;
        zz = (xx+limitbv)-limitbv2;
        carry = zz*invlimit;
        // Compute the carry on next word
        x[IDX(j)] = xx - zz;
        // And the balanced remainder in current one
    }
    g_carry[threadID]=carry;
    if(g_err[threadID] < maxerr)
        g_err[threadID] = maxerr;
}

//      Functions that do the normalization of the result of a multiplication or squaring
__global__ void cuda_rnormalize2_kernel(
    // Used for rational bases DWFFT
    double  *x,
    int     N,
    int     error_log,
    int     noadd,
    int     nomul,
    double  limitbv,
    double  limitbv2,
    double  invlimit,
    int     STRIDE,
    double  bv,
    double  bv2,
    double  *g_carry,
    float   *g_err,
    int     MULBYCONST,
    double  SMALLMULCONST,
    int     addinindex,
    double  addinvalue,
    int     cg      // 25/03/21
)
{
    register int    	j;
    register double         xx, zz, tx;
    register double         carry = 0.0;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

    if(threadID*STRIDE < N && threadID*STRIDE+STRIDE >= N)  {
        carry = g_carry[threadID];
        if (carry)  {
            j = 0;
//            if (plus) 
//                carry = -carry;
            carry *= -cg;    // accept abs(c)!=1 here! 25/03/21
            while (carry)   {       
                tx = x[IDX(j)];
                xx = tx + carry;
                zz = (xx+limitbv)-limitbv2;
                carry = zz*invlimit;
                // Compute the carry on next word
                x[IDX(j)] = xx - zz;
                // And the balanced remainder in current word
                if (++j == N)   {
                    j = 0;
//                    if (plus)				   carry = -carry;
                    carry *= -cg;    // accept abs(c)!=1 here! 25/03/21
                }
            }
        }
    }
    else    {
        carry = g_carry[threadID];
        j = threadID*STRIDE+STRIDE;
        while (carry)   {
            tx = x[IDX(j)];
            xx = tx + carry;
            zz = (xx+limitbv)-limitbv2;
            carry = zz*invlimit;
            // Compute the carry on next word
            x[IDX(j)] = xx - zz;
            // And the balanced remainder in current word
            j++;
        }
    }
}

double
rnormalize( // Used for rational bases DWFFT
    double  *x,
    int     N,
    int     error_log,
    int     noadd,
    int     nomul
)
{
    register int    j;
    register double *px = x, xx, zz, bv = BIGVAL2, invlimit = limit_inverse_high;// JP 08/07/17
    register double carry = 0.0, limitbv = (limit_high*BIGVAL2)-BIGVAL2, err, maxerr = 0.0;
                // JP 08/07/17

    if (!noadd)
        x[addinindex] += addinvalue;
        // Add the optional small constant
    for (j=0; j<N; ++j) {
        if (MULBYCONST && !nomul)
            *px *= SMALLMULCONST;
        // Optionaly multiply by a small constant
        xx = (*px + bv) - bv;
        // Round to the nearest integer
        if (error_log ) {
        // Compute the rounding error if required
            if (xx<0)
                err = fabs(-xx + *px);  
            else 
                err = fabs(xx  - *px);
            if (err > maxerr) 
                maxerr = err;
        }
        xx += carry;
        zz = (xx+limitbv)-limitbv;
        carry = zz*invlimit;
        // Compute the carry on next word
        *(px++) = xx - zz;
        // And the balanced remainder in current word
    }
    if (carry)  {
        j = 0;
        px = x;
//        if (plus)
//            carry = -carry;
        carry *= -cg;    // accept abs(c)!=1 here! 25/03/21
        while (carry)   {
            xx = *px + carry;
            zz = (xx+limitbv)-limitbv;
            carry = zz*invlimit;
            // Compute the carry on next word
            *(px++) = xx - zz;
            // And the balanced remainder in current word
            if (++j == N)   {
                j = 0;
                px = x;
//                if (plus)
//                    carry = -carry;
                carry *= -cg;    // accept abs(c)!=1 here! 25/03/21
            }
        }
    }
    return(maxerr);
}

// Modular reduction of a zero padded integer

/*
__global__ void cuda_modred_kernel (
    double *s,
    double *scral,
    int len,
    int hwcount,
    int lwcount,
    int hindex,
    double bv,
    double limit_high,
    double mult,
    double invmult,
    double g_limitbv
    
)
{
        const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
	register int 	i, j, liminf;
	register double carry = 0.0, xx, yy, zz, q;
        
        i = threadID;
        j = threadID + len -1;
        liminf = threadID + len -hindex;
        if (i<12)
            scral[i] = carry;   // Zero 12 double words in scratch area
            
}*/

void
modred (
    double *x
)
{
    register double *pscr = scral;
    // Init. ; Point to scratch area
    register double *px = x+FFTLEN-1;
    // Point to last FFT word
    register long hindex = hwcount;
    // Count of upper FFT words
    register double carry = 0.0, bv = BIGVAL2, limitbv = (limit_high*bv)-bv, xx, yy, zz, q;

    if (tdebug)
        gwypstart_timer (3);
    // Zero 12 double words in scratch area
    *pscr++ = carry;			
    *pscr++ = carry;
    *pscr++ = carry;
    *pscr++ = carry;
    *pscr++ = carry;
    *pscr++ = carry;
    *pscr++ = carry;
    *pscr++ = carry;
    *pscr++ = carry;
    *pscr++ = carry;
    *pscr++ = carry;
    *pscr++ = carry;

    while (hindex-- > 0) {// divide by mult the upper FFT words and save them in scratch area
        xx = *px+carry*limit_high;
        q = (xx*invmult+bv)-bv;
                            // q = xx/mult integer
        carry = xx-mult*q;  // carry = xx%h
        *px-- = 0.0;        // zero the high word
        *pscr++ = shift*q;
                        // save -inc*q*2^(ng-q)%bits
    }
    px++;
    pscr--;
    *px = carry;
    px = x;
    hindex = lwcount;
    carry = bv;
    while (hindex-- > 0) {
    // Add or subtract saved words into the lower part of FFT
        xx = *px + *pscr-- + carry;			// xx = x + *pscr + carry
        yy = xx+limitbv;				// y = xx/limit_high*limit_high + bv
        zz = yy-limitbv;				// z = xx/limit_high*limit_high
        carry = limit_inverse_high*yy;
        // carry = y / limit_high + bv
        *px++ = xx-zz;					// new x = xx%limit_high
    }
    px = x + hwoffset + 4;				// Point to 5th upper FFT word;
    carry = *px;
    *px-- = 0.0;
    xx = *px+carry*limit_high;
    q = (xx*invmult+bv)-bv;				// q = xx/mult integer
    carry = xx-mult*q;
    *px-- = 0.0;
    *pscr = shift*q;
    xx = *px+carry*limit_high;
    q = (xx*invmult+bv)-bv;				// q = xx/mult integer
    carry = xx-mult*q;
    *px-- = 0.0;
    *(pscr+1) = shift*q;
    xx = *px + carry*limit_high;
    q = (xx*invmult+bv)-bv;			
    // q = xx/mult integer
    carry = xx-mult*q;
    *px-- = 0.0;   // added
    *(pscr+2) = shift*q;
    xx = *px + carry*limit_high;
    q = (xx*invmult+bv)-bv;				// q = xx/mult integer
    carry = xx-mult*q;
    *px = 0.0;	// added
    *(pscr+3) = shift*q;	
    while ((carry>(limit_high/2)) || (carry<(-limit_high/2))) {
        xx = carry+bv;  // Split the remainder
        yy = xx + limitbv;				// y = xx/limit_high*limit_high + bv
        zz = yy - limitbv;				// z = xx/limit_high*limit_high
        carry = limit_inverse_high*yy-bv;
        // carry = y / limit_high
        *px++ = xx-zz;	// Save lower bits
    }
    *(px) = carry;
    px = x; // Reload source pointer
    xx = *px + *(pscr+3) + bv;
    yy = xx + limitbv;					// y = xx/limit_high*limit_high + bv
    zz = yy - limitbv;					// z = xx/limit_high*limit_high
    carry = limit_inverse_high*yy;
    // carry = y / limit_high
    *px++ = xx-zz;  // Save new value
    xx = *px + *(pscr+2) + carry;
    yy = xx + limitbv;					// y = xx/limit_high*limit_high + bv
    zz = yy - limitbv;					// z = xx/limit_high*limit_high
    carry = limit_inverse_high*yy;
    // carry = y / limit_high
    *px++ = xx-zz;  // Save new value
    xx = *px + *(pscr+1) + carry;
    yy = xx + limitbv;					// y = xx/limit_high*limit_high + bv
    zz = yy - limitbv;					// z = xx/limit_high*limit_high
    carry = limit_inverse_high*yy;
    // carry = y / limit_high
    *px++ = xx-zz;  // Save new value
    xx = *px + *(pscr+0) + carry;
    yy = xx + limitbv;					// y = xx/limit_high*limit_high + bv
    zz = yy - limitbv;					// z = xx/limit_high*limit_high
    carry = limit_inverse_high*yy;
    // carry = y / limit_high
    *px++ = xx-zz;  // Save new value
    xx = *px + carry;
    yy = xx + limitbv;					// y = xx/limit_high*limit_high + bv
    zz = yy - limitbv;					// z = xx/limit_high*limit_high
    carry = limit_inverse_high*yy;
    // carry = y / limit_high
    *px++ = xx-zz;  // Save new value
    *px	+= carry - bv;  // Adjust final carry
    if (tdebug)
        gwypend_timer (3);
}

int
check_balanced(	// Check if the balanced form of a result is correct
    double *x,
    int N
)
{
    int j;
    double lim, *ptrx = x;

    for (j=0; j<N; ++j) {
        lim = hlimit[j];
//        assert ((*ptrx<=lim) && (*ptrx>=-lim));
        if ((*ptrx>lim) || (*ptrx<-lim))
            return (TRUE);  // ERROR
        ++ptrx;
    }
    return FALSE;           // OK
}

__global__ void
cuda_check_balanced_kernel(
// Check if the balanced form of a result is correct
    double *x,
    double *hlimit,
    int N,
    int *balerr
)
{
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

    if (threadID<N)
        assert ((x[threadID]<=hlimit[threadID]) && (x[threadID]>=-hlimit[threadID]));
//        *balerr |= ((x[threadID]>hlimit[threadID]) || (x[threadID]<-hlimit[threadID]));
}

__global__ void
cuda_geterr_kernel(
    float   *err,
    float   *g_err,
    int     N
)
{
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (threadID<N && *err<g_err[threadID])
        *err=g_err[threadID];
}

__global__ void
cuda_mulbyconst_kernel(
    double  *out,
    double  *in,
    double  c,
    int     n
)
{
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    if(threadID<n)
        out[threadID]=in[threadID]*c;
}

__global__ void
cuda_mul_kernel(
    double  *out,
    double  *in,
    double  *in2,
    int     n
)
{
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    if(threadID<n)
        out[threadID]=in[threadID]*in2[threadID];
}

__global__ void
cuda_mul_two_to_phi_kernel(
    double  *x,
    double  *cxin,
    double  *two_to_phi,
    int     hn
)
{
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    if(threadID<hn) {
        cxin[threadID*2]=x[threadID]*two_to_phi[threadID];
        cxin[threadID*2+1]=x[threadID+hn]*two_to_phi[threadID+hn];
    }
}

__global__ void
cuda_fold_kernel(
    double  *x,
    double  *cxin,
    int     hn
)
{
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    if(threadID<hn) {
        cxin[threadID*2]=x[threadID];
        cxin[threadID*2+1]=x[threadID+hn];
    }
}

__global__ void
cuda_mul_two_to_minusphi_kernel(
    double  *x,
    double  *cxin,
    double  *two_to_minusphi,
    int     hn
)
{
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    if(threadID<hn) {
        x[threadID]=cxin[threadID*2]*two_to_minusphi[threadID];
        x[threadID+hn]=cxin[threadID*2+1]*two_to_minusphi[threadID+hn];
    }
}

__global__ void
cuda_unfold_kernel(
    double  *x,
    double  *cxin,
    double  c,
    int     hn
)
{
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    if(threadID<hn) {
        x[threadID]=cxin[threadID*2]*c;
        x[threadID+hn]=cxin[threadID*2+1]*c;
    }
}

//	Functions that do squaring and multiplications of large integers ; inputs and output are normalized

double
cuda_lucas_square(
    // Squaring of a large integer ; input and output normalized
    double  *x,
    int     N,
    int     error_log,
    int     noadd,
    int     nomul,
    int     flag
)
{
    register int    j, hn = N/2;
    float           err;
    int             STRIDE;
    STRIDE=STRIDE_DIM;
    err=0;
    dim3 grid(STRIDE_DIM/BLOCK_DIM,STRIDE_DIM/BLOCK_DIM, 1);
    dim3 threads(BLOCK_DIM, BLOCK_DIM, 1);
    if (tdebug)
        gwypstart_timer (5);
    if (zp || generic) {
        if(compl2)  {
            if(flag&1)  {
                cutilSafeCall(hipMemcpy(cuda_x,x,sizeof(double)*N,hipMemcpyHostToDevice));
            }
            cuda_fold_kernel<<<(hn+127)/128,128>>>(cuda_x,cuda_cxin,hn);
            cuda_fftwsquare_g(N);   // DWT squaring
            cuda_unfold_kernel<<<(hn+127)/128,128>>>(cuda_x,cuda_cxin,(double)ttmp,hn);
            for(j=0;j<N;j+=(STRIDE_DIM*STRIDE_DIM))
                transpose<<<grid, threads>>>((double *)&cuda_tmp[j],(double *)&cuda_x[j],(int)  STRIDE_DIM,(int) STRIDE_DIM);
            cuda_rnormalize_kernel<<<(N+STRIDE*128-1)/STRIDE/128,128>>>(cuda_tmp, N, error_log, noadd, nomul,limit_high_bigval,limit_high_bigval,limit_inverse_high,STRIDE,BIGVAL2,BIGVAL2,g_carry,g_err,MULBYCONST,SMALLMULCONST,addinindex,addinvalue,cg);    // 25/03/21
            cuda_rnormalize2_kernel<<<(N+STRIDE*128-1)/STRIDE/128,128>>>(cuda_tmp, N, error_log, noadd, nomul,limit_high_bigval,limit_high_bigval,limit_inverse_high,STRIDE,BIGVAL2,BIGVAL2,g_carry,g_err,MULBYCONST,SMALLMULCONST,addinindex,addinvalue,cg);   // 25/03/21
            for(j=0;j<N;j+=(STRIDE_DIM*STRIDE_DIM))
                transpose<<<grid, threads>>>((double *)&cuda_x[j],(double *)&cuda_tmp[j],(int)  STRIDE_DIM,(int) STRIDE_DIM);
            if (error_log) {
                cuda_check_balanced_kernel <<<(N+128-1)/128,128>>>(cuda_x, g_hlimit, N, &balerr);
                cutilSafeCall(hipMemcpy(l_err,g_err,sizeof(float)*(N+STRIDE-1)/STRIDE,hipMemcpyDeviceToHost));
                for(j=0;j<(N+STRIDE-1)/STRIDE;j++)
                    if(err<l_err[j])
                        err=l_err[j];
//                    cuda_geterr_kernel <<<(N+STRIDE*128-1)/STRIDE/128,128>>>(&err, g_err, (N+STRIDE-1)/STRIDE);
            }
            if(/*!error_log && */flag&2)    {
                cutilSafeCall(hipMemcpy(x,cuda_x,sizeof(double)*N,hipMemcpyDeviceToHost));
            } 
            if (tdebug)
                gwypend_timer (5);
            if (!generic)   {
                if (tdebug)
                    gwypstart_timer (3);
                modred (x);
                if (tdebug)
                    gwypend_timer (3);
            }
            return(err);
        }
        else    {
            if(flag&1)  {
                cutilSafeCall(hipMemcpy(cuda_xin,x,sizeof(double)*N,hipMemcpyHostToDevice));
            }
            cuda_fftwsquare_g(N);   // DWT squaring
            for(j=0;j<N;j+=(STRIDE_DIM*STRIDE_DIM))
                mul_const_transpose<<<grid, threads>>>((double *)&cuda_tmp[j],(double *)&cuda_xin[j],(double)ttmp,(int)  STRIDE_DIM,(int) STRIDE_DIM);
            cuda_rnormalize_kernel<<<(N+STRIDE*128-1)/STRIDE/128,128>>>(cuda_tmp, N, error_log, noadd, nomul,limit_high_bigval,limit_high_bigval,limit_inverse_high,STRIDE,BIGVAL2,BIGVAL2,g_carry,g_err,MULBYCONST,SMALLMULCONST,addinindex,addinvalue,cg);
            cuda_rnormalize2_kernel<<<(N+STRIDE*128-1)/STRIDE/128,128>>>(cuda_tmp, N, error_log, noadd, nomul,limit_high_bigval,limit_high_bigval,limit_inverse_high,STRIDE,BIGVAL2,BIGVAL2,g_carry,g_err,MULBYCONST,SMALLMULCONST,addinindex,addinvalue,cg);
            for(j=0;j<N;j+=(STRIDE_DIM*STRIDE_DIM))
                    transpose<<<grid, threads>>>((double *)&cuda_x[j],(double *)&cuda_tmp[j],(int) STRIDE_DIM,(int) STRIDE_DIM);
            if (error_log)  {
                cutilSafeCall(hipMemcpy(l_err,g_err,sizeof(float)*(N+STRIDE-1)/STRIDE,hipMemcpyDeviceToHost));
                for(j=0;j<(N+STRIDE-1)/STRIDE;j++)
                    if(err<l_err[j])
                        err=l_err[j];
                cuda_check_balanced_kernel <<<(N+128-1)/128,128>>>(cuda_x, g_hlimit, N, &balerr);
            }
            if(/*!error_log && */flag&2) {
                cutilSafeCall(hipMemcpy(x,cuda_x,sizeof(double)*N,hipMemcpyDeviceToHost));
            } 
            for(j=0;j<N;j+=(STRIDE_DIM*STRIDE_DIM))
        		transpose<<<grid, threads>>>((double *)&cuda_xin[j],(double *)&cuda_tmp[j],(int) STRIDE_DIM,(int) STRIDE_DIM);
            if (tdebug)
                gwypend_timer (5);
            if (!generic)   {
                if (tdebug)
                    gwypstart_timer (3);
                modred (x);
                if (tdebug)
                    gwypend_timer (3);
            }

            return(err);
        }
    }
    else
        if(compl2)  {
            if(flag&1)  {
                cutilSafeCall(hipMemcpy(cuda_x,x,sizeof(double)*N,hipMemcpyHostToDevice));
            }
            cuda_mul_two_to_phi_kernel<<<(hn+127)/128,128>>>(cuda_x,cuda_cxin,cuda_two_to_phi,hn);
            cuda_fftwsquare_g(N);   // DWT squaring
            cuda_mul_two_to_minusphi_kernel<<<(hn+127)/128,128>>>(cuda_x,cuda_cxin,cuda_two_to_minusphi,hn);
            for(j=0;j<N;j+=(STRIDE_DIM*STRIDE_DIM))
                transpose<<<grid, threads>>>((double *)&cuda_tmp[j],(double *)&cuda_x[j],(int)  STRIDE_DIM,(int) STRIDE_DIM);
            cuda_inormalize_kernel<<<(N+STRIDE*128-1)/STRIDE/128,128>>>(cuda_tmp, N, error_log, noadd, nomul,g_limitbv,g_limitbv,g_invlimit,STRIDE,BIGVAL2,BIGVAL2,g_carry,g_err,wrapindex,wrapfactor,MULBYCONST,SMALLMULCONST,addinindex,addinvalue,cg);
            cuda_inormalize2_kernel<<<(N+STRIDE*128-1)/STRIDE/128,128>>>(cuda_tmp, N, error_log, noadd, nomul,g_limitbv,g_limitbv,g_invlimit,STRIDE,BIGVAL2,BIGVAL2,g_carry,g_err,wrapindex,wrapfactor,MULBYCONST,SMALLMULCONST,addinindex,addinvalue,cg);
            for(j=0;j<N;j+=(STRIDE_DIM*STRIDE_DIM))
                transpose<<<grid, threads>>>((double *)&cuda_x[j],(double *)&cuda_tmp[j],(int)  STRIDE_DIM,(int) STRIDE_DIM);
        	if (error_log)
            {
            cuda_check_balanced_kernel <<<(N+128-1)/128,128>>>(cuda_x, g_hlimit, N, &balerr);
            cutilSafeCall(hipMemcpy(l_err,g_err,sizeof(float)*(N+STRIDE-1)/STRIDE,hipMemcpyDeviceToHost));
            for(j=0;j<(N+STRIDE-1)/STRIDE;j++)
                if(err<l_err[j])
                    err=l_err[j];
//                    cuda_geterr_kernel <<<(N+STRIDE*128-1)/STRIDE/128,128>>>(&err, g_err, (N+STRIDE-1)/STRIDE);
        }
        if(/*!error_log && */flag&2)    {
            cutilSafeCall(hipMemcpy(x,cuda_x,sizeof(double)*N,hipMemcpyDeviceToHost));
        } 
        if (tdebug)
            gwypend_timer (5);
        return(err);
    }
    else    {
        if(flag&1)  {
            cutilSafeCall(hipMemcpy(cuda_x,x,sizeof(double)*N,hipMemcpyHostToDevice));
            cuda_mul_kernel<<<(N+127)/128,128>>>(cuda_xin,cuda_x,cuda_two_to_phi, N);
        }
        cuda_fftwsquare_g(N);	// DWT squaring
        for(j=0;j<N;j+=(STRIDE_DIM*STRIDE_DIM))
            mul_0_transpose<<<grid, threads>>>((double *)&cuda_tmp[j],(double *)&cuda_xin[j],(double *)&cuda_two_to_minusphi[j],(int)  STRIDE_DIM,(int) STRIDE_DIM);
        cuda_inormalize_kernel<<<(N+STRIDE*128-1)/STRIDE/128,128>>>(cuda_tmp, N, error_log, noadd, nomul,g_limitbv,g_limitbv,g_invlimit,STRIDE,BIGVAL2,BIGVAL2,g_carry,g_err,wrapindex,wrapfactor,MULBYCONST,SMALLMULCONST,addinindex,addinvalue,cg);
        cuda_inormalize2_kernel<<<(N+STRIDE*128-1)/STRIDE/128,128>>>(cuda_tmp, N, error_log, noadd, nomul,g_limitbv,g_limitbv,g_invlimit,STRIDE,BIGVAL2,BIGVAL2,g_carry,g_err,wrapindex,wrapfactor,MULBYCONST,SMALLMULCONST,addinindex,addinvalue,cg);
        for(j=0;j<N;j+=(STRIDE_DIM*STRIDE_DIM))
            transpose<<<grid, threads>>>((double *)&cuda_x[j],(double *)&cuda_tmp[j],(int) STRIDE_DIM,(int) STRIDE_DIM);
        if (error_log)  {
            cutilSafeCall(hipMemcpy(l_err,g_err,sizeof(float)*(N+STRIDE-1)/STRIDE,hipMemcpyDeviceToHost));
            for(j=0;j<(N+STRIDE-1)/STRIDE;j++)
                if(err<l_err[j])
                    err=l_err[j];
            cuda_check_balanced_kernel <<<(N+128-1)/128,128>>>(cuda_x, g_hlimit, N, &balerr);
        }
        if(/*!error_log && */flag&2) {
            cutilSafeCall(hipMemcpy(x,cuda_x,sizeof(double)*N,hipMemcpyDeviceToHost));
        } 
        for(j=0;j<N;j+=(STRIDE_DIM*STRIDE_DIM))
            mul_1_transpose<<<grid, threads>>>((double *)&cuda_xin[j],(double *)&cuda_tmp[j],(double *)&cuda_two_to_phi[j],(int) STRIDE_DIM,(int) STRIDE_DIM);
        if (tdebug)
            gwypend_timer (5);
        return(err);
    }
}

double
cuda_lucas_square_generic(
    // Squaring of a large integer ; input and output normalized, generic reduction included.
    double  *x,
    int     N,
    int     error_log,
    int     noadd,
    int     nomul,
    int     flag
)
{
    register int    j, hn = N/2;
    register double err;
    int             STRIDE;
    
    STRIDE=STRIDE_DIM;
    err=0;
    dim3 grid(STRIDE_DIM/BLOCK_DIM,STRIDE_DIM/BLOCK_DIM, 1);
    dim3 threads(BLOCK_DIM, BLOCK_DIM, 1);
    if (tdebug)
        gwypstart_timer (5);
    if(compl2)  {
        if(flag&1)  {
            cutilSafeCall(hipMemcpy(cuda_x,x,sizeof(double)*N,hipMemcpyHostToDevice));
        }
        cuda_fold_kernel<<<(hn+127)/128,128>>>(cuda_x,cuda_cxin,hn);
        cuda_fftwsquare_g(N);	// DWT squaring
        cuda_unfold_kernel<<<(hn+127)/128,128>>>(cuda_x,cuda_cxin,(double)ttmp,hn);
        for(j=0;j<N;j+=(STRIDE_DIM*STRIDE_DIM))
            transpose<<<grid, threads>>>((double *)&cuda_tmp[j],(double *)&cuda_x[j],(int)  STRIDE_DIM,(int) STRIDE_DIM);
        cuda_rnormalize_kernel<<<(N+STRIDE*128-1)/STRIDE/128,128>>>(cuda_tmp, N, error_log, noadd, nomul,limit_high_bigval,limit_high_bigval,limit_inverse_high,STRIDE,BIGVAL2,BIGVAL2,g_carry,g_err,MULBYCONST,SMALLMULCONST,addinindex,addinvalue,cg);
        cuda_rnormalize2_kernel<<<(N+STRIDE*128-1)/STRIDE/128,128>>>(cuda_tmp, N, error_log, noadd, nomul,limit_high_bigval,limit_high_bigval,limit_inverse_high,STRIDE,BIGVAL2,BIGVAL2,g_carry,g_err,MULBYCONST,SMALLMULCONST,addinindex,addinvalue,cg);
        for(j=0;j<N;j+=(STRIDE_DIM*STRIDE_DIM))
            transpose<<<grid, threads>>>((double *)&cuda_x[j],(double *)&cuda_tmp[j],(int)  STRIDE_DIM,(int) STRIDE_DIM);
        if (error_log)  {
            cuda_check_balanced_kernel <<<(N+128-1)/128,128>>>(cuda_x, g_hlimit, N, &balerr);
            cutilSafeCall(hipMemcpy(l_err,g_err,sizeof(float)*(N+STRIDE-1)/STRIDE,hipMemcpyDeviceToHost));
            for(j=0;j<(N+STRIDE-1)/STRIDE;j++)
                if(err<l_err[j])
                    err=l_err[j];
        }
        cuda_gwypcopyzero_kernel<<<(N+127)/128,128>>>(cuda_x, cuda_tmp_g, zerowordslow, FFTLEN);
        cuda_fold_kernel<<<(hn+127)/128,128>>>(cuda_tmp_g,cuda_cyin,hn);
        cuda_fftwmulbyr_g(N);
        cuda_unfold_kernel<<<(hn+127)/128,128>>>(cuda_tmp_g,cuda_cyin,(double)ttmp,hn);
        for(j=0;j<N;j+=(STRIDE_DIM*STRIDE_DIM))
            transpose<<<grid, threads>>>((double *)&cuda_tmp[j],(double *)&cuda_tmp_g[j],(int)  STRIDE_DIM,(int) STRIDE_DIM);
        cuda_rnormalize_kernel<<<(N+STRIDE*128-1)/STRIDE/128,128>>>(cuda_tmp, N, error_log, 1, 1,limit_high_bigval,limit_high_bigval,limit_inverse_high,STRIDE,BIGVAL2,BIGVAL2,g_carry,g_err,MULBYCONST,SMALLMULCONST,addinindex,addinvalue,cg);
        cuda_rnormalize2_kernel<<<(N+STRIDE*128-1)/STRIDE/128,128>>>(cuda_tmp, N, error_log, 1, 1,limit_high_bigval,limit_high_bigval,limit_inverse_high,STRIDE,BIGVAL2,BIGVAL2,g_carry,g_err,MULBYCONST,SMALLMULCONST,addinindex,addinvalue,cg);
        for(j=0;j<N;j+=(STRIDE_DIM*STRIDE_DIM))
            transpose<<<grid, threads>>>((double *)&cuda_tmp_g[j],(double *)&cuda_tmp[j],(int)  STRIDE_DIM,(int) STRIDE_DIM);
        if (error_log)  {
            cuda_check_balanced_kernel <<<(N+128-1)/128,128>>>(cuda_tmp_g, g_hlimit, N, &balerr);
            cutilSafeCall(hipMemcpy(l_err,g_err,sizeof(float)*(N+STRIDE-1)/STRIDE,hipMemcpyDeviceToHost));
            for(j=0;j<(N+STRIDE-1)/STRIDE;j++)
                if(err<l_err[j])
                    err=l_err[j];
        }
        cuda_gwypsetzero_kernel<<<(N+127)/128,128>>>(cuda_tmp_g, zerowordshigh, FFTLEN);
        cuda_fold_kernel<<<(hn+127)/128,128>>>(cuda_tmp_g,cuda_cyin,hn);
        cuda_fftwmulbym_g(N);
        cuda_unfold_kernel<<<(hn+127)/128,128>>>(cuda_tmp_g,cuda_cyin,(double)ttmp,hn);
        for(j=0;j<N;j+=(STRIDE_DIM*STRIDE_DIM))
            transpose<<<grid, threads>>>((double *)&cuda_tmp[j],(double *)&cuda_tmp_g[j],(int)  STRIDE_DIM,(int) STRIDE_DIM);
        cuda_rnormalize_kernel<<<(N+STRIDE*128-1)/STRIDE/128,128>>>(cuda_tmp, N, error_log, 1, 1,limit_high_bigval,limit_high_bigval,limit_inverse_high,STRIDE,BIGVAL2,BIGVAL2,g_carry,g_err,MULBYCONST,SMALLMULCONST,addinindex,addinvalue,cg);
        cuda_rnormalize2_kernel<<<(N+STRIDE*128-1)/STRIDE/128,128>>>(cuda_tmp, N, error_log, 1, 1,limit_high_bigval,limit_high_bigval,limit_inverse_high,STRIDE,BIGVAL2,BIGVAL2,g_carry,g_err,MULBYCONST,SMALLMULCONST,addinindex,addinvalue,cg);
        for(j=0;j<N;j+=(STRIDE_DIM*STRIDE_DIM))
            transpose<<<grid, threads>>>((double *)&cuda_tmp_g[j],(double *)&cuda_tmp[j],(int)  STRIDE_DIM,(int) STRIDE_DIM);
        if (error_log)  {
            cuda_check_balanced_kernel <<<(N+128-1)/128,128>>>(cuda_tmp_g, g_hlimit, N, &balerr);
            cutilSafeCall(hipMemcpy(l_err,g_err,sizeof(float)*(N+STRIDE-1)/STRIDE,hipMemcpyDeviceToHost));
            for(j=0;j<(N+STRIDE-1)/STRIDE;j++)
                if(err<l_err[j])
                    err=l_err[j];
        }
        cuda_gwypaddquick_kernel<<<(N+127)/128,128>>>(cuda_tmp_g, cuda_x, FFTLEN);
        cuda_gwypsubquick_kernel<<<(N+127)/128,128>>>(cuda_m, cuda_x, FFTLEN);
        if(flag&2)
            cutilSafeCall(hipMemcpy(x,cuda_x,sizeof(double)*N,hipMemcpyDeviceToHost));
        if (tdebug)
            gwypend_timer (5);
        return(err);
    }
    else    {
        if(flag&1)  {
            cutilSafeCall(hipMemcpy(cuda_xin,x,sizeof(double)*N,hipMemcpyHostToDevice));
        }
        cuda_fftwsquare_g(N);	// DWT squaring
        for(j=0;j<N;j+=(STRIDE_DIM*STRIDE_DIM))
            mul_const_transpose<<<grid, threads>>>((double *)&cuda_tmp[j],(double *)&cuda_xin[j],(double)ttmp,(int)  STRIDE_DIM,(int) STRIDE_DIM);
        cuda_rnormalize_kernel<<<(N+STRIDE*128-1)/STRIDE/128,128>>>(cuda_tmp, N, error_log, noadd, nomul,limit_high_bigval,limit_high_bigval,limit_inverse_high,STRIDE,BIGVAL2,BIGVAL2,g_carry,g_err,MULBYCONST,SMALLMULCONST,addinindex,addinvalue,cg);
        cuda_rnormalize2_kernel<<<(N+STRIDE*128-1)/STRIDE/128,128>>>(cuda_tmp, N, error_log, noadd, nomul,limit_high_bigval,limit_high_bigval,limit_inverse_high,STRIDE,BIGVAL2,BIGVAL2,g_carry,g_err,MULBYCONST,SMALLMULCONST,addinindex,addinvalue,cg);
        for(j=0;j<N;j+=(STRIDE_DIM*STRIDE_DIM))
            transpose<<<grid, threads>>>((double *)&cuda_x[j],(double *)&cuda_tmp[j],(int) STRIDE_DIM,(int) STRIDE_DIM);
        if (error_log)  {
            cuda_check_balanced_kernel <<<(N+128-1)/128,128>>>(cuda_x, g_hlimit, N, &balerr);
            cutilSafeCall(hipMemcpy(l_err,g_err,sizeof(float)*(N+STRIDE-1)/STRIDE,hipMemcpyDeviceToHost));
            for(j=0;j<(N+STRIDE-1)/STRIDE;j++)
                if(err<l_err[j])
                    err=l_err[j];
        }
        cuda_gwypcopyzero_kernel<<<(N+127)/128,128>>>(cuda_x, cuda_yin, zerowordslow, FFTLEN);
        cuda_fftwmulbyr_g(N);
        for(j=0;j<N;j+=(STRIDE_DIM*STRIDE_DIM))
            mul_const_transpose<<<grid, threads>>>((double *)&cuda_tmp[j],(double *)&cuda_yin[j],(double)ttmp,(int)  STRIDE_DIM,(int) STRIDE_DIM);
        cuda_rnormalize_kernel<<<(N+STRIDE*128-1)/STRIDE/128,128>>>(cuda_tmp, N, error_log, 1, 1,limit_high_bigval,limit_high_bigval,limit_inverse_high,STRIDE,BIGVAL2,BIGVAL2,g_carry,g_err,MULBYCONST,SMALLMULCONST,addinindex,addinvalue,cg);
        cuda_rnormalize2_kernel<<<(N+STRIDE*128-1)/STRIDE/128,128>>>(cuda_tmp, N, error_log, 1, 1,limit_high_bigval,limit_high_bigval,limit_inverse_high,STRIDE,BIGVAL2,BIGVAL2,g_carry,g_err,MULBYCONST,SMALLMULCONST,addinindex,addinvalue,cg);
        for(j=0;j<N;j+=(STRIDE_DIM*STRIDE_DIM))
            transpose<<<grid, threads>>>((double *)&cuda_yin[j],(double *)&cuda_tmp[j],(int) STRIDE_DIM,(int) STRIDE_DIM);
        if (error_log)  {
            cuda_check_balanced_kernel <<<(N+128-1)/128,128>>>(cuda_yin, g_hlimit, N, &balerr);
            cutilSafeCall(hipMemcpy(l_err,g_err,sizeof(float)*(N+STRIDE-1)/STRIDE,hipMemcpyDeviceToHost));
            for(j=0;j<(N+STRIDE-1)/STRIDE;j++)
                if(err<l_err[j])
                    err=l_err[j];
        }
        cuda_gwypsetzero_kernel<<<(N+127)/128,128>>>(cuda_yin, zerowordshigh, FFTLEN);
        cuda_fftwmulbym_g(N);
        for(j=0;j<N;j+=(STRIDE_DIM*STRIDE_DIM))
            mul_const_transpose<<<grid, threads>>>((double *)&cuda_tmp[j],(double *)&cuda_yin[j],(double)ttmp,(int)  STRIDE_DIM,(int) STRIDE_DIM);
        cuda_rnormalize_kernel<<<(N+STRIDE*128-1)/STRIDE/128,128>>>(cuda_tmp, N, error_log, 1, 1,limit_high_bigval,limit_high_bigval,limit_inverse_high,STRIDE,BIGVAL2,BIGVAL2,g_carry,g_err,MULBYCONST,SMALLMULCONST,addinindex,addinvalue,cg);
        cuda_rnormalize2_kernel<<<(N+STRIDE*128-1)/STRIDE/128,128>>>(cuda_tmp, N, error_log, 1, 1,limit_high_bigval,limit_high_bigval,limit_inverse_high,STRIDE,BIGVAL2,BIGVAL2,g_carry,g_err,MULBYCONST,SMALLMULCONST,addinindex,addinvalue,cg);
        for(j=0;j<N;j+=(STRIDE_DIM*STRIDE_DIM))
            transpose<<<grid, threads>>>((double *)&cuda_tmp_g[j],(double *)&cuda_tmp[j],(int) STRIDE_DIM,(int) STRIDE_DIM);
        if (error_log)  {
            cuda_check_balanced_kernel <<<(N+128-1)/128,128>>>(cuda_tmp_g, g_hlimit, N, &balerr);
            cutilSafeCall(hipMemcpy(l_err,g_err,sizeof(float)*(N+STRIDE-1)/STRIDE,hipMemcpyDeviceToHost));
            for(j=0;j<(N+STRIDE-1)/STRIDE;j++)
                if(err<l_err[j])
                    err=l_err[j];
        }
        cuda_gwypsubquick_kernel<<<(N+127)/128,128>>>(cuda_tmp_g, cuda_x, FFTLEN);
        if(flag&2)
            cutilSafeCall(hipMemcpy(x,cuda_x,sizeof(double)*N,hipMemcpyDeviceToHost));
        if (tdebug)
            gwypend_timer (5);
        return(err);
    }
}

//	Funtions that do squaring and multiplications of large integers ; inputs and output are normalized

double
lucas_square(	// Squaring of a large integer ; input and output normalized
    double  *x,
    int     N,
    int     error_log,
    int     noadd,
    int     nomul
)
{
    register int        j, hn = N/2;
    register double     err;

    if (tdebug)
        gwypstart_timer (5);
    if (zp || generic)
        if (compl2) {   // Transform an N sized real FFT array into an N/2 sized complex FFT one
            for (j=0; j<hn; ++j)    {
                cxin[j][0] = x[j];
                cxin[j][1] = x[j+hn];
            }
        }
        else {
            for (j=0; j<N; ++j)
                xin[j] = x[j];
        }
    else
        if (compl2) {	// Transform an N sized real FFT array into an N/2 sized complex FFT one
            for (j=0; j<hn; ++j)    {
                cxin[j][0] = x[j] * two_to_phi[j];
                cxin[j][1] = x[j+hn] * two_to_phi[j+hn];
            }
        }
        else {
            for (j=0; j<N; ++j)
                xin[j] = x[j] * two_to_phi[j];
        }
    fftwsquare_g(N);   // DWT squaring
    if (zp || generic)
        if (compl2) {   // Unfold the N/2 sized complex ouput array into a N sized real one
            for (j=0; j<hn; ++j)    {
                    x[j] = cxin[j][0] * ttmp;
                    x[j+hn] = cxin[j][1] * ttmp;
            }
        }
        else {
            for (j=0; j<N; ++j)
                x[j] = xin[j] * ttmp;
        }
    else
        if (compl2) {   // Unfold the N/2 sized complex ouput array into a N sized real one
            for (j=0; j<hn; ++j)    {
                x[j] = cxin[j][0] *  two_to_minusphi[j];
                x[j+hn] = cxin[j][1] *  two_to_minusphi[j+hn];
            }
        }
        else {
            for (j=0; j<N; ++j)
                x[j] = xin[j] *  two_to_minusphi[j];
        }
    if (tdebug)
        gwypend_timer (5);
    if (tdebug)
        gwypstart_timer (2);
    err = (zp || generic)? rnormalize(x, N, error_log, noadd, nomul) : inormalize(x, N, error_log, noadd, nomul);
    if (tdebug)
        gwypend_timer (2);
    if (zp)    {
        if (tdebug)
            gwypstart_timer (3);
        modred (x);
        if (tdebug)
            gwypend_timer (3);
    }
    if (error_log)
        balerr = check_balanced(x, N);
    return(err);
}

double
cuda_lucas_mul(	// Multiplication of large integers ; inputs and output normalized
    double  *x,
    double  *y,
    int     N,
    int     error_log,
    int     noadd,
    int     nomul,
    int     flag
)
{
    register int    j, hn = N/2;
    register double err;
    int             STRIDE;
    
    STRIDE=STRIDE_DIM;
    err=0;
    dim3 grid(STRIDE_DIM/BLOCK_DIM,STRIDE_DIM/BLOCK_DIM, 1);
    dim3 threads(BLOCK_DIM, BLOCK_DIM, 1);
    if (zp || generic)
        if(compl2)  {
            if (tdebug)
                gwypstart_timer (5);
            if(flag&1)  {
                cutilSafeCall(hipMemcpy(cuda_x,x,sizeof(double)*N,hipMemcpyHostToDevice));
                cutilSafeCall(hipMemcpy(cuda_y,y,sizeof(double)*N,hipMemcpyHostToDevice));
            }
            cuda_fold_kernel<<<(hn+127)/128,128>>>(cuda_x,cuda_cxin,hn);
            cuda_fold_kernel<<<(hn+127)/128,128>>>(cuda_y,cuda_cyin,hn);
            cuda_fftwmul_g(N);	// DWT multipication
            cuda_unfold_kernel<<<(hn+127)/128,128>>>(cuda_y,cuda_cyin,(double)ttmp,hn);
            for(j=0;j<N;j+=(STRIDE_DIM*STRIDE_DIM))
                transpose<<<grid, threads>>>((double *)&cuda_tmp[j],(double *)&cuda_y[j],(int)  STRIDE_DIM,(int) STRIDE_DIM);
            cuda_rnormalize_kernel<<<(N+STRIDE*128-1)/STRIDE/128,128>>>(cuda_tmp, N, error_log, noadd, nomul,limit_high_bigval,limit_high_bigval,limit_inverse_high,STRIDE,BIGVAL2,BIGVAL2,g_carry,g_err,MULBYCONST,SMALLMULCONST,addinindex,addinvalue,cg);
            cuda_rnormalize2_kernel<<<(N+STRIDE*128-1)/STRIDE/128,128>>>(cuda_tmp, N, error_log, noadd, nomul,limit_high_bigval,limit_high_bigval,limit_inverse_high,STRIDE,BIGVAL2,BIGVAL2,g_carry,g_err,MULBYCONST,SMALLMULCONST,addinindex,addinvalue,cg);
            for(j=0;j<N;j+=(STRIDE_DIM*STRIDE_DIM))
                transpose<<<grid, threads>>>((double *)&cuda_y[j],(double *)&cuda_tmp[j],(int)  STRIDE_DIM,(int) STRIDE_DIM);
            cuda_check_balanced_kernel <<<(N+128-1)/128,128>>>(cuda_y, g_hlimit, N, &balerr);
            cutilSafeCall(hipMemcpy(l_err,g_err,sizeof(float)*(N+STRIDE-1)/STRIDE,hipMemcpyDeviceToHost));
            for(j=0;j<(N+STRIDE-1)/STRIDE;j++)
                if(err<l_err[j])
                    err=l_err[j];
            if(/*!error_log && */flag&2)
                cutilSafeCall(hipMemcpy(y,cuda_y,sizeof(double)*N,hipMemcpyDeviceToHost));
            if (tdebug)
                gwypend_timer (5);
            if (!generic)   {
                if (tdebug)
                    gwypstart_timer (3);
                modred (y);
                if (tdebug)
                    gwypend_timer (3);
            }
            return(err);
        }
        else    {
            if (tdebug)
                gwypstart_timer (5);
            if(flag&1)  {
                cutilSafeCall(hipMemcpy(cuda_xin,x,sizeof(double)*N,hipMemcpyHostToDevice));
                cutilSafeCall(hipMemcpy(cuda_yin,y,sizeof(double)*N,hipMemcpyHostToDevice));
            }
            cuda_fftwmul_g(N);// DWT multiplication
            for(j=0;j<N;j+=(STRIDE_DIM*STRIDE_DIM))
                mul_const_transpose<<<grid, threads>>>((double *)&cuda_tmp[j],(double *)&cuda_yin[j],(double)ttmp,(int)  STRIDE_DIM,(int) STRIDE_DIM);
            cuda_rnormalize_kernel<<<(N+STRIDE*128-1)/STRIDE/128,128>>>(cuda_tmp, N, error_log, noadd, nomul,limit_high_bigval,limit_high_bigval,limit_inverse_high,STRIDE,BIGVAL2,BIGVAL2,g_carry,g_err,MULBYCONST,SMALLMULCONST,addinindex,addinvalue,cg);
            cuda_rnormalize2_kernel<<<(N+STRIDE*128-1)/STRIDE/128,128>>>(cuda_tmp, N, error_log, noadd, nomul,limit_high_bigval,limit_high_bigval,limit_inverse_high,STRIDE,BIGVAL2,BIGVAL2,g_carry,g_err,MULBYCONST,SMALLMULCONST,addinindex,addinvalue,cg);
            for(j=0;j<N;j+=(STRIDE_DIM*STRIDE_DIM))
                transpose<<<grid, threads>>>((double *)&cuda_y[j],(double *)&cuda_tmp[j],(int) STRIDE_DIM,(int) STRIDE_DIM);
            if (error_log)    {
                cutilSafeCall(hipMemcpy(l_err,g_err,sizeof(float)*(N+STRIDE-1)/STRIDE,hipMemcpyDeviceToHost));
                for(j=0;j<(N+STRIDE-1)/STRIDE;j++)
                    if(err<l_err[j])
                        err=l_err[j];
                cuda_check_balanced_kernel <<<(N+128-1)/128,128>>>(cuda_y, g_hlimit, N, &balerr);
            }
            if(/*!error_log && */flag&2) {
                cutilSafeCall(hipMemcpy(y,cuda_y,sizeof(double)*N,hipMemcpyDeviceToHost));
            } 
            if (tdebug)
                gwypend_timer (5);
            if (!generic)   {
                if (tdebug)
                    gwypstart_timer (3);
                modred (y);
                if (tdebug)
                    gwypend_timer (3);
            }
            return(err);
        }
    else
        if(compl2)  {
            if (tdebug)
                gwypstart_timer (5);
            if(flag&1){
                cutilSafeCall(hipMemcpy(cuda_x,x,sizeof(double)*N,hipMemcpyHostToDevice));
                cutilSafeCall(hipMemcpy(cuda_y,y,sizeof(double)*N,hipMemcpyHostToDevice));
            }
            cuda_mul_two_to_phi_kernel<<<(hn+127)/128,128>>>(cuda_x,cuda_cxin,cuda_two_to_phi,hn);
            cuda_mul_two_to_phi_kernel<<<(hn+127)/128,128>>>(cuda_y,cuda_cyin,cuda_two_to_phi,hn);
            cuda_fftwmul_g(N);	// DWT multipication
            cuda_mul_two_to_minusphi_kernel<<<(hn+127)/128,128>>>(cuda_y,cuda_cyin,cuda_two_to_minusphi,hn);
            for(j=0;j<N;j+=(STRIDE_DIM*STRIDE_DIM))
                transpose<<<grid, threads>>>((double *)&cuda_tmp[j],(double *)&cuda_y[j],(int)  STRIDE_DIM,(int) STRIDE_DIM);
            cuda_inormalize_kernel<<<(N+STRIDE*128-1)/STRIDE/128,128>>>(cuda_tmp, N, error_log, noadd, nomul,g_limitbv,g_limitbv,g_invlimit,STRIDE,BIGVAL2,BIGVAL2,g_carry,g_err,wrapindex,wrapfactor,MULBYCONST,SMALLMULCONST,addinindex,addinvalue,cg);
            cuda_inormalize2_kernel<<<(N+STRIDE*128-1)/STRIDE/128,128>>>(cuda_tmp, N, error_log, noadd, nomul,g_limitbv,g_limitbv,g_invlimit,STRIDE,BIGVAL2,BIGVAL2,g_carry,g_err,wrapindex,wrapfactor,MULBYCONST,SMALLMULCONST,addinindex,addinvalue,cg);
            for(j=0;j<N;j+=(STRIDE_DIM*STRIDE_DIM))
                transpose<<<grid, threads>>>((double *)&cuda_y[j],(double *)&cuda_tmp[j],(int)  STRIDE_DIM,(int) STRIDE_DIM);
            if (error_log) {
                cuda_check_balanced_kernel <<<(N+128-1)/128,128>>>(cuda_y, g_hlimit, N, &balerr);
                cutilSafeCall(hipMemcpy(l_err,g_err,sizeof(float)*(N+STRIDE-1)/STRIDE,hipMemcpyDeviceToHost));
                for(j=0;j<(N+STRIDE-1)/STRIDE;j++)
                    if(err<l_err[j])
                        err=l_err[j];
            }
            if(/*!error_log && */flag&2)    {
                cutilSafeCall(hipMemcpy(y,cuda_y,sizeof(double)*N,hipMemcpyDeviceToHost));
            } 
            if (tdebug)
                gwypend_timer (5);
            return(err);
        }
        else    {
            if (tdebug)
                gwypstart_timer (5);
            if(flag&1)  {
                cutilSafeCall(hipMemcpy(cuda_x,x,sizeof(double)*N,hipMemcpyHostToDevice));
                cutilSafeCall(hipMemcpy(cuda_y,y,sizeof(double)*N,hipMemcpyHostToDevice));
            }
            cuda_mul_kernel<<<(N+127)/128,128>>>(cuda_xin,cuda_x,cuda_two_to_phi, N);
            cuda_mul_kernel<<<(N+127)/128,128>>>(cuda_yin,cuda_y,cuda_two_to_phi, N);
            cuda_fftwmul_g(N);// DWT multiplication
            for(j=0;j<N;j+=(STRIDE_DIM*STRIDE_DIM))
                mul_0_transpose<<<grid, threads>>>((double *)&cuda_tmp[j],(double *)&cuda_yin[j],(double *)&cuda_two_to_minusphi[j],(int)  STRIDE_DIM,(int) STRIDE_DIM);
            cuda_inormalize_kernel<<<(N+STRIDE*128-1)/STRIDE/128,128>>>(cuda_tmp, N, error_log, noadd, nomul,g_limitbv,g_limitbv,g_invlimit,STRIDE,BIGVAL2,BIGVAL2,g_carry,g_err,wrapindex,wrapfactor,MULBYCONST,SMALLMULCONST,addinindex,addinvalue,cg);
            cuda_inormalize2_kernel<<<(N+STRIDE*128-1)/STRIDE/128,128>>>(cuda_tmp, N, error_log, noadd, nomul,g_limitbv,g_limitbv,g_invlimit,STRIDE,BIGVAL2,BIGVAL2,g_carry,g_err,wrapindex,wrapfactor,MULBYCONST,SMALLMULCONST,addinindex,addinvalue,cg
);
            for(j=0;j<N;j+=(STRIDE_DIM*STRIDE_DIM))
                transpose<<<grid, threads>>>((double *)&cuda_y[j],(double *)&cuda_tmp[j],(int) STRIDE_DIM,(int) STRIDE_DIM);
            if (error_log)    {
                cutilSafeCall(hipMemcpy(l_err,g_err,sizeof(float)*(N+STRIDE-1)/STRIDE,hipMemcpyDeviceToHost));
                for(j=0;j<(N+STRIDE-1)/STRIDE;j++)
                    if(err<l_err[j])
                        err=l_err[j];
                cuda_check_balanced_kernel <<<(N+128-1)/128,128>>>(cuda_y, g_hlimit, N, &balerr);
            }
            if(/*!error_log && */flag&2)
                cutilSafeCall(hipMemcpy(y,cuda_y,sizeof(double)*N,hipMemcpyDeviceToHost));
            for(j=0;j<N;j+=(STRIDE_DIM*STRIDE_DIM))
                mul_1_transpose<<<grid, threads>>>((double *)&cuda_yin[j],(double *)&cuda_tmp[j],(double *)&cuda_two_to_phi[j],(int) STRIDE_DIM,(int) STRIDE_DIM);
            if (tdebug)
                gwypend_timer (5);
            return(err);
	}
}

double
cuda_lucas_mul_generic(	// Multiplication of large integers ; inputs and output normalized
    double  *x,
    double  *y,
    int     N,
    int     error_log,
    int     noadd,
    int     nomul,
    int     flag
)
{
    register int 		j, hn = N/2;
    register double 	err;
    int STRIDE;

    STRIDE=STRIDE_DIM;
    err=0;
    dim3 grid(STRIDE_DIM/BLOCK_DIM,STRIDE_DIM/BLOCK_DIM, 1);
    dim3 threads(BLOCK_DIM, BLOCK_DIM, 1);

    if (tdebug)
        gwypstart_timer (5);
    if(compl2)  {
        if(flag&1){
            cutilSafeCall(hipMemcpy(cuda_x,x,sizeof(double)*N,hipMemcpyHostToDevice));
            cutilSafeCall(hipMemcpy(cuda_y,y,sizeof(double)*N,hipMemcpyHostToDevice));
        }
        cuda_fold_kernel<<<(hn+127)/128,128>>>(cuda_x,cuda_cxin,hn);
        cuda_fold_kernel<<<(hn+127)/128,128>>>(cuda_y,cuda_cyin,hn);
        cuda_fftwmul_g(N);  // DWT multipication
        cuda_unfold_kernel<<<(hn+127)/128,128>>>(cuda_y,cuda_cyin,(double)ttmp,hn);
        for(j=0;j<N;j+=(STRIDE_DIM*STRIDE_DIM))
            transpose<<<grid, threads>>>((double *)&cuda_tmp[j],(double *)&cuda_y[j],(int)  STRIDE_DIM,(int) STRIDE_DIM);
        cuda_rnormalize_kernel<<<(N+STRIDE*128-1)/STRIDE/128,128>>>(cuda_tmp, N, error_log, noadd, nomul,limit_high_bigval,limit_high_bigval,limit_inverse_high,STRIDE,BIGVAL2,BIGVAL2,g_carry,g_err,MULBYCONST,SMALLMULCONST,addinindex,addinvalue,cg);
        cuda_rnormalize2_kernel<<<(N+STRIDE*128-1)/STRIDE/128,128>>>(cuda_tmp, N, error_log, noadd, nomul,limit_high_bigval,limit_high_bigval,limit_inverse_high,STRIDE,BIGVAL2,BIGVAL2,g_carry,g_err,MULBYCONST,SMALLMULCONST,addinindex,addinvalue,cg);
        for(j=0;j<N;j+=(STRIDE_DIM*STRIDE_DIM))
            transpose<<<grid, threads>>>((double *)&cuda_y[j],(double *)&cuda_tmp[j],(int)  STRIDE_DIM,(int) STRIDE_DIM);
        cuda_gwypcopyzero_kernel<<<(N+127)/128,128>>>(cuda_y, cuda_tmp_g, zerowordslow, FFTLEN);
        cuda_fold_kernel<<<(hn+127)/128,128>>>(cuda_tmp_g,cuda_cyin,hn);
        cuda_fftwmulbyr_g(N);
        cuda_unfold_kernel<<<(hn+127)/128,128>>>(cuda_tmp_g,cuda_cyin,(double)ttmp,hn);
        for(j=0;j<N;j+=(STRIDE_DIM*STRIDE_DIM))
            transpose<<<grid, threads>>>((double *)&cuda_tmp[j],(double *)&cuda_tmp_g[j],(int)  STRIDE_DIM,(int) STRIDE_DIM);
        cuda_rnormalize_kernel<<<(N+STRIDE*128-1)/STRIDE/128,128>>>(cuda_tmp, N, error_log, 1, 1,limit_high_bigval,limit_high_bigval,limit_inverse_high,STRIDE,BIGVAL2,BIGVAL2,g_carry,g_err,MULBYCONST,SMALLMULCONST,addinindex,addinvalue,cg);
        cuda_rnormalize2_kernel<<<(N+STRIDE*128-1)/STRIDE/128,128>>>(cuda_tmp, N, error_log, 1, 1,limit_high_bigval,limit_high_bigval,limit_inverse_high,STRIDE,BIGVAL2,BIGVAL2,g_carry,g_err,MULBYCONST,SMALLMULCONST,addinindex,addinvalue,cg);
        for(j=0;j<N;j+=(STRIDE_DIM*STRIDE_DIM))
            transpose<<<grid, threads>>>((double *)&cuda_tmp_g[j],(double *)&cuda_tmp[j],(int)  STRIDE_DIM,(int) STRIDE_DIM);
        if (error_log) {
            cuda_check_balanced_kernel <<<(N+128-1)/128,128>>>(cuda_tmp_g, g_hlimit, N, &balerr);
            cutilSafeCall(hipMemcpy(l_err,g_err,sizeof(float)*(N+STRIDE-1)/STRIDE,hipMemcpyDeviceToHost));
            for(j=0;j<(N+STRIDE-1)/STRIDE;j++)
                if(err<l_err[j])
                    err=l_err[j];
        }
        cuda_gwypsetzero_kernel<<<(N+127)/128,128>>>(cuda_tmp_g, zerowordshigh, FFTLEN);
        cuda_fold_kernel<<<(hn+127)/128,128>>>(cuda_tmp_g,cuda_cyin,hn);
        cuda_fftwmulbym_g(N);
        cuda_unfold_kernel<<<(hn+127)/128,128>>>(cuda_tmp_g,cuda_cyin,(double)ttmp,hn);
        for(j=0;j<N;j+=(STRIDE_DIM*STRIDE_DIM))
            transpose<<<grid, threads>>>((double *)&cuda_tmp[j],(double *)&cuda_tmp_g[j],(int)  STRIDE_DIM,(int) STRIDE_DIM);
        cuda_rnormalize_kernel<<<(N+STRIDE*128-1)/STRIDE/128,128>>>(cuda_tmp, N, error_log, 1, 1,limit_high_bigval,limit_high_bigval,limit_inverse_high,STRIDE,BIGVAL2,BIGVAL2,g_carry,g_err,MULBYCONST,SMALLMULCONST,addinindex,addinvalue,cg);
        cuda_rnormalize2_kernel<<<(N+STRIDE*128-1)/STRIDE/128,128>>>(cuda_tmp, N, error_log, 1, 1,limit_high_bigval,limit_high_bigval,limit_inverse_high,STRIDE,BIGVAL2,BIGVAL2,g_carry,g_err,MULBYCONST,SMALLMULCONST,addinindex,addinvalue,cg);
        for(j=0;j<N;j+=(STRIDE_DIM*STRIDE_DIM))
            transpose<<<grid, threads>>>((double *)&cuda_tmp_g[j],(double *)&cuda_tmp[j],(int)  STRIDE_DIM,(int) STRIDE_DIM);
        if (error_log)  {
            cuda_check_balanced_kernel <<<(N+128-1)/128,128>>>(cuda_tmp_g, g_hlimit, N, &balerr);
            cutilSafeCall(hipMemcpy(l_err,g_err,sizeof(float)*(N+STRIDE-1)/STRIDE,hipMemcpyDeviceToHost));
            for(j=0;j<(N+STRIDE-1)/STRIDE;j++)
                if(err<l_err[j])
                    err=l_err[j];
        }
        cuda_gwypaddquick_kernel<<<(N+127)/128,128>>>(cuda_tmp_g, cuda_y, FFTLEN);
        cuda_gwypsubquick_kernel<<<(N+127)/128,128>>>(cuda_m, cuda_y, FFTLEN);
        if (flag&2)
            cutilSafeCall(hipMemcpy(y,cuda_y,sizeof(double)*N,hipMemcpyDeviceToHost));
        if (tdebug)
            gwypend_timer (5);
        return(err);
    }
    else    {
        if(flag&1){
            cutilSafeCall(hipMemcpy(cuda_xin,x,sizeof(double)*N,hipMemcpyHostToDevice));
            cutilSafeCall(hipMemcpy(cuda_yin,y,sizeof(double)*N,hipMemcpyHostToDevice));
        }
        cuda_fftwmul_g(N);  // DWT multiplication
        for(j=0;j<N;j+=(STRIDE_DIM*STRIDE_DIM))
            mul_const_transpose<<<grid, threads>>>((double *)&cuda_tmp[j],(double *)&cuda_yin[j],(double)ttmp,(int)  STRIDE_DIM,(int) STRIDE_DIM);
        cuda_rnormalize_kernel<<<(N+STRIDE*128-1)/STRIDE/128,128>>>(cuda_tmp, N, error_log, noadd, nomul,limit_high_bigval,limit_high_bigval,limit_inverse_high,STRIDE,BIGVAL2,BIGVAL2,g_carry,g_err,MULBYCONST,SMALLMULCONST,addinindex,addinvalue,cg);
        cuda_rnormalize2_kernel<<<(N+STRIDE*128-1)/STRIDE/128,128>>>(cuda_tmp, N, error_log, noadd, nomul,limit_high_bigval,limit_high_bigval,limit_inverse_high,STRIDE,BIGVAL2,BIGVAL2,g_carry,g_err,MULBYCONST,SMALLMULCONST,addinindex,addinvalue,cg);
        for(j=0;j<N;j+=(STRIDE_DIM*STRIDE_DIM))
            transpose<<<grid, threads>>>((double *)&cuda_y[j],(double *)&cuda_tmp[j],(int) STRIDE_DIM,(int) STRIDE_DIM);
        if (error_log)  {
            cuda_check_balanced_kernel <<<(N+128-1)/128,128>>>(cuda_y, g_hlimit, N, &balerr);
            cutilSafeCall(hipMemcpy(l_err,g_err,sizeof(float)*(N+STRIDE-1)/STRIDE,hipMemcpyDeviceToHost));
            for(j=0;j<(N+STRIDE-1)/STRIDE;j++)
                if(err<l_err[j])
                    err=l_err[j];
        }
        cuda_gwypcopyzero_kernel<<<(N+127)/128,128>>>(cuda_y, cuda_yin, zerowordslow, FFTLEN);
        cuda_fftwmulbyr_g(N);
        for(j=0;j<N;j+=(STRIDE_DIM*STRIDE_DIM))
            mul_const_transpose<<<grid, threads>>>((double *)&cuda_tmp[j],(double *)&cuda_yin[j],(double)ttmp,(int)  STRIDE_DIM,(int) STRIDE_DIM);
        cuda_rnormalize_kernel<<<(N+STRIDE*128-1)/STRIDE/128,128>>>(cuda_tmp, N, error_log, 1, 1,limit_high_bigval,limit_high_bigval,limit_inverse_high,STRIDE,BIGVAL2,BIGVAL2,g_carry,g_err,MULBYCONST,SMALLMULCONST,addinindex,addinvalue,cg);
        cuda_rnormalize2_kernel<<<(N+STRIDE*128-1)/STRIDE/128,128>>>(cuda_tmp, N, error_log, 1, 1,limit_high_bigval,limit_high_bigval,limit_inverse_high,STRIDE,BIGVAL2,BIGVAL2,g_carry,g_err,MULBYCONST,SMALLMULCONST,addinindex,addinvalue,cg);
        for(j=0;j<N;j+=(STRIDE_DIM*STRIDE_DIM))
            transpose<<<grid, threads>>>((double *)&cuda_yin[j],(double *)&cuda_tmp[j],(int) STRIDE_DIM,(int) STRIDE_DIM);
        if (error_log){
            cuda_check_balanced_kernel <<<(N+128-1)/128,128>>>(cuda_yin, g_hlimit, N, &balerr);
            cutilSafeCall(hipMemcpy(l_err,g_err,sizeof(float)*(N+STRIDE-1)/STRIDE,hipMemcpyDeviceToHost));
            for(j=0;j<(N+STRIDE-1)/STRIDE;j++)
                if(err<l_err[j])
                    err=l_err[j];
        }
        cuda_gwypsetzero_kernel<<<(N+127)/128,128>>>(cuda_yin, zerowordshigh, FFTLEN);
        cuda_fftwmulbym_g(N);
        for(j=0;j<N;j+=(STRIDE_DIM*STRIDE_DIM))
            mul_const_transpose<<<grid, threads>>>((double *)&cuda_tmp[j],(double *)&cuda_yin[j],(double)ttmp,(int)  STRIDE_DIM,(int) STRIDE_DIM);
        cuda_rnormalize_kernel<<<(N+STRIDE*128-1)/STRIDE/128,128>>>(cuda_tmp, N, error_log, 1, 1,limit_high_bigval,limit_high_bigval,limit_inverse_high,STRIDE,BIGVAL2,BIGVAL2,g_carry,g_err,MULBYCONST,SMALLMULCONST,addinindex,addinvalue,cg);
        cuda_rnormalize2_kernel<<<(N+STRIDE*128-1)/STRIDE/128,128>>>(cuda_tmp, N, error_log, 1, 1,limit_high_bigval,limit_high_bigval,limit_inverse_high,STRIDE,BIGVAL2,BIGVAL2,g_carry,g_err,MULBYCONST,SMALLMULCONST,addinindex,addinvalue,cg);
        for(j=0;j<N;j+=(STRIDE_DIM*STRIDE_DIM))
            transpose<<<grid, threads>>>((double *)&cuda_tmp_g[j],(double *)&cuda_tmp[j],(int) STRIDE_DIM,(int) STRIDE_DIM);
        if (error_log){
            cuda_check_balanced_kernel <<<(N+128-1)/128,128>>>(cuda_tmp_g, g_hlimit, N, &balerr);
            cutilSafeCall(hipMemcpy(l_err,g_err,sizeof(float)*(N+STRIDE-1)/STRIDE,hipMemcpyDeviceToHost));
            for(j=0;j<(N+STRIDE-1)/STRIDE;j++)
                if(err<l_err[j])
                    err=l_err[j];
        }
        cuda_gwypsubquick_kernel<<<(N+127)/128,128>>>(cuda_tmp_g, cuda_y, FFTLEN);
        if(flag&2)
            cutilSafeCall(hipMemcpy(y,cuda_y,sizeof(double)*N,hipMemcpyDeviceToHost));
        if (tdebug)
            gwypend_timer (5);
        return(err);
    }
}

double
lucas_mul(	// Multiplication of large integers ; inputs and output normalized
    double  *x,
    double  *y,
    int     N,
    int     error_log,
    int     noadd,
    int     nomul
)
{
    register int    j, hn = N/2;
    register double err;
    if (tdebug)
        gwypstart_timer (5);
    if (zp || generic)
        if (compl2) {	// Transform N sized real FFT arrays into N/2 sized complex FFT ones
            for (j=0; j<hn; ++j)    {
                cxin[j][0] = x[j];
                cxin[j][1] = x[j+hn];
                cyin[j][0] = y[j];
                cyin[j][1] = y[j+hn];
            }
        }
        else {
            for (j=0; j<N; ++j) {
                xin[j] = x[j];
                yin[j] = y[j];
            }
        }
    else
        if (compl2) {	// Transform N sized real FFT arrays into N/2 sized complex FFT ones
            for (j=0; j<hn; ++j)    {
                cxin[j][0] = x[j] * two_to_phi[j];
                cxin[j][1] = x[j+hn] * two_to_phi[j+hn];
                cyin[j][0] = y[j] * two_to_phi[j];
                cyin[j][1] = y[j+hn] * two_to_phi[j+hn];
            }
        }
        else {
            for (j=0; j<N; ++j) {
                xin[j] = x[j] * two_to_phi[j];
                yin[j] = y[j] * two_to_phi[j];
            }
        }
    fftwmul_g(N);   // DWT mutiplication
    if (zp || generic)
        if (compl2) {	// Unfold the N/2 sized complex ouput array into a N sized real one
            for (j=0; j<hn; ++j)    {
                y[j] = cyin[j][0] * ttmp;
                y[j+hn] = cyin[j][1] * ttmp;
            }
        }
        else {
            for (j=0; j<N; ++j)
                y[j] = yin[j] * ttmp;
        }
    else
        if (compl2) {	// Unfold the N/2 sized complex ouput array into a N sized real one
            for (j=0; j<hn; ++j)    {
                y[j] = cyin[j][0] *  two_to_minusphi[j];
                y[j+hn] = cyin[j][1] *  two_to_minusphi[j+hn];
            }
        }
        else {
            for (j=0; j<N; ++j)
                y[j] = yin[j] *  two_to_minusphi[j];
        }
    if (tdebug)
        gwypend_timer (5);
    if (tdebug)
        gwypstart_timer (2);
    err = (zp || generic)? rnormalize(y, N, error_log, noadd, nomul) : inormalize(y, N, error_log, noadd, nomul);
    if (tdebug)
        gwypend_timer (2);
    if (zp)    {
        if (tdebug)
            gwypstart_timer (3);
        modred (y);
        if (tdebug)
            gwypend_timer (3);
    }
    if (error_log)
        balerr = check_balanced (y, N);
    return(err);
}

/* --------- Initialization routines ---------- */


int set_fftlen (double k, unsigned long b, unsigned long n, signed long c) {
    double bpw, bpwg, tbpw, tbpwg, kbits = 0.0, rdwt, rzpad, rgeneric; 
    double bpddwt, bpdzpad, bpdgeneric;
    double log2k = 0.0, log2b, bitsmc = log(MAXMULCONST)/log(2.0);
    int incfft, fftwincr, fftdwt = 2, fftzpad = 2, fftgeneric = 2 , zpad = 0;
    unsigned long len, ki = (unsigned long)k;
    unsigned long safety_bits;
    int i;

    printfunction = (verbose)? both_output : screen_output;
    log2b = log2(b);

    while (ki) {    // Compute bit lengh of k
        kbits += 1.0;
        ki >>= 1;
    }
    
    if (k > 0.0) {
        log2k = log2(k);
    }
    len = bitlen (gmodulus);
    rdwt = len;     // Exponent for DWT mode
    rzpad = len + len + 32;
    
// Exponents for zero padded and generic modes
    
    rgeneric = rzpad + 2*EB;
    
// Compute the fftlength for each mode

    i=0;    // Compute generic fftlengh (raw).
    incfft = FFTINC;
    while ((xgwtablep[i].max_exp) != 0) {
        fftgeneric = xgwtablep[i].fftlen;
        if (fftgeneric < STRIDE_DIM*2)
            goto next1;
        else if ((double) n * log2b / (double) fftgeneric > 26.0)
            goto next1;
        else if (rgeneric >= xgwtablep[i].max_exp)
            goto next1;
        else {
            if (incfft) {
                i += incfft;
                incfft =0;
                continue;
            }
            break;
        }
next1:
        i++;
    }
    bpdgeneric = (double) (b*xgwtablep[i].max_exp)/(double) fftgeneric;
    if (bpdgeneric > MAXBPD)
        bpdgeneric = MAXBPD;
    incfft = FFTINC;
    fftwincr = fftgeneric / 16;
    while (1) {  // Compute generic fftlengh (fine).
        bpwg = RINT((rgeneric+fftgeneric-1)/fftgeneric);
        tbpwg = 2.0*bpwg + bitsmc + 2.0;
//        if (tbpwg <= MAXBITSPERDOUBLE)
        if (tbpwg <= bpdgeneric)
            break;
        else
            fftgeneric += fftwincr;
    }
    while (incfft-- > 0)
        fftgeneric += fftwincr;
    
    incfft = FFTINC;
    i=0;    // Compute zero padded fftlengh (raw).
    while ((/*max_exp = */xgwtablep[i].max_exp) != 0) {
        fftzpad = xgwtablep[i].fftlen;
        if (fftzpad < STRIDE_DIM*2)
            goto next2;
        else if ((double) n * log2b / (double) fftzpad > 26.0)
            goto next2;
        else if (rzpad >= xgwtablep[i].max_exp)
            goto next2;
        else {
            if (incfft) {
                i += incfft;
                incfft =0;
                continue;
            }
            break;
        }
next2:
        i++;
    }
    bpdzpad = (double) (b*xgwtablep[i].max_exp)/(double) fftzpad;
    if (bpdzpad > MAXBPD)
        bpdzpad = MAXBPD;
    incfft = FFTINC;
    // Compute zero padded FFT length (fine)
    fftwincr = fftzpad / 16;
    while (1) {
        bpw = RINT((rzpad+fftzpad-1)/fftzpad);
        tbpw = 2.0*bpw + bitsmc;
//        if (tbpw <= MAXBITSPERDOUBLE)
        if (tbpw <= bpdzpad)
            break;
        else
            fftzpad += fftwincr;
    }
    while (incfft-- > 0)
        fftzpad += fftwincr;
    
    i=0;    // Compute IBDWT FFT length (raw)
    incfft = FFTINC;
    while ((/*max_exp = */xgwtable[i].max_exp) != 0) {
        fftdwt = xgwtable[i].fftlen;
        bpw = RINT((rdwt+fftdwt-1)/fftdwt) + 0.25*log2(abs(c)); // 15/03/21
        tbpw = 2.0*bpw + kbits + bitsmc + 0.6*log((double)fftdwt)/log(2.0);
        if (fftdwt < STRIDE_DIM*2)
            goto next3;
        else if ((double) n * log2b / (double) fftdwt > 26.0)
            goto next3;
        else if (rdwt >= xgwtable[i].max_exp)
            goto next3;
            // Top carry adjust can only handle k values of 34 bits or less.        
        else if (log2k >= 34.0)
            goto next3;
//        else if (tbpw > MAXBITSPERDOUBLE)   // 29/03/21
//            goto next3;
        else {
            if (incfft) {
                i += incfft;
                incfft =0;
                continue;
            }
            break;
        }
next3:
        i++;
    }            
    bpddwt = (double) (b*xgwtable[i].max_exp)/(double) fftdwt;
    if (bpddwt > MAXBPD)
        bpddwt = MAXBPD;
    incfft = FFTINC;
    fftwincr = fftdwt / 16;
    while (fftdwt <= 33554432) {
        bpw = RINT((rdwt+fftdwt-1)/fftdwt) + 0.25*log2(abs(c)); // 15/03/21
        tbpw = 2.0*bpw + kbits + bitsmc + 0.6*log((double)fftdwt)/log(2.0);
//        if (tbpw <= MAXBITSPERDOUBLE)
        if (tbpw <= bpddwt)
            break;
        else
            fftdwt += fftwincr;
    }
/*    sprintf (gwypbuf, "maxbpw = %g\n",
            bpddwt);
            if (printfunction != NULL)
                (*printfunction)(gwypbuf); */
    while (incfft-- > 0)
        fftdwt += (fftwincr);
    if (!generic && !zpad && (tbpw > bpddwt))     // IBDWT impossible... 27/03/21
        generic = 1;
    if (generic) {
        zpad = 0;
        FFTLEN = fftgeneric;
        bpwg = RINT((rgeneric+fftgeneric-1)/fftgeneric);
        kg = 1.0;
        bg = 2;
        ng = (unsigned long)bpwg*FFTLEN;
        cg = (zcomplex)? 1 : -1;
            // Optional, defaulted to TRUE.
        grecip = newgiant (((unsigned long)rgeneric) + 16);  //  JP 12/01/21 conversion was missing!
                            // 07/02/21 grecip seems to require very much memory...
        nb_malloc++;
        itog (1, grecip); 
        gshiftleft (len + len + EB, grecip); 
        divg (gmodulus, grecip);// compute len+EB+1 bits of reciprocal 
        gshiftleft (ng - len - len - EB, grecip);
        // shift so gwmul routines wrap quotient to lower end of fft 
        zerowordslow = (len - EB) / (unsigned long)bpwg; 
        zerowordshigh = FFTLEN - (len + (unsigned long)bpwg - 1) / ((unsigned long)bpwg - 1); 
	avg_num_b_per_word = ((zpad || generic) ? n * 2.0 : (logb(k) + n)) / FFTLEN;
	safety_bits = len - (unsigned long) ((double) zerowordslow * avg_num_b_per_word) + 3;        
        /* Calculate the maximum allowable size of a number used as input */
        /* to gwmul.  We will make sure gwsmallmul does not generate any */
        /* results bigger than this. */
	GW_GEN_MOD_MAX = (unsigned long) floor ((double)((n/2-safety_bits+len-8)/2) / avg_num_b_per_word);
        modulus = gwypalloc ();
        nb_malloc++;
        recip = gwypalloc ();
        nb_malloc++;
        gwyptmp = gwypalloc ();
        nb_malloc++;
        gianttogwyp (gmodulus, modulus);
        gianttogwyp (grecip, recip);
    }
    else if (zpad || (kbits > MAXKBITS)) {
        zpad = 1;
        FFTLEN = fftzpad;
        bpw = RINT((rzpad+fftzpad-1)/fftzpad);
        kg = 1.0;
        bg = 2;
        ng = (unsigned long)bpw*FFTLEN;
        cg = (zcomplex)? 1 : -1;
            // Optional, defaulted to TRUE.
        // init. the modred code.
	avg_num_b_per_word = ((zpad || generic) ? ng * 2.0 : (logb(k) + ng)) / FFTLEN;
        inc = c;    // copy for zp
        bits = ng/FFTLEN;
        limit_high = (double)(1<<bits);
        limit_inverse_high = 1.0/limit_high;
        limit_high_bigval = (limit_high*BIGVAL2)-BIGVAL2;
        mult = k;   // copy for zp
        invmult = 1.0/mult;
        rem = (ng - n)%bits;
        shift = (double) (1L << rem) * -inc;
        hwcount = (ng - n)/bits;
        lwcount = (((ng - n)/bits + 9)/8)*8;
        hwoffset = (n + bits - 1) / bits;
        if (debug) {
            sprintf (gwypbuf, "bits = %lu, ng = %lu, hwcount = %lu, lwcount = %lu, hwoffset = %lu\n", bits, ng, hwcount, lwcount, hwoffset);
            if (printfunction != NULL)
                (*printfunction)(gwypbuf);
            sprintf (gwypbuf, "limit_high = %g, limit_inverse_high = %g, limit_high_bigval = %g, mult = %g, invmult = %g, rem = %lu, shift = %g\n",
            limit_high, limit_inverse_high, limit_high_bigval, mult, invmult, rem, shift);
            if (printfunction != NULL)
                (*printfunction)(gwypbuf);
        }
        temp = (long) malloc ((hwcount + 24) * sizeof (double) + 256);
        nb_malloc++;
        scr = (double *) temp;
            // address of scratch area
        scral = (double *) ((temp + 7) & 0xFFFFFFFFFFFFFFF8);// double word aligned
    }
    else {  // IBDWT
        zpad = 0;
        bpw = RINT((rdwt+fftdwt-1)/fftdwt);
        FFTLEN = fftdwt;
//	}       Use the code below only for IBDWT, JP 08/07/17
	
// 	{       And not for rational base !
        int ii,jj;
        if(g_fftlen != 0)
            FFTLEN = g_fftlen * 2 + 2 ;
        for(jj = ii = 64;ii < (FFTLEN >> 1) ;ii*=2)
            for(jj=ii;jj < (FFTLEN >> 1) && jj < ii*2 ;jj+=ii/4);
        FFTLEN = jj;
        g_fftlen = jj;
        {
            if(g_fftlen != 0)
                FFTLEN = g_fftlen * 2 + 2 ;
            for(jj = ii = 64;ii < (FFTLEN >> 1) ;ii*=2)
                for(jj=ii;jj < (FFTLEN >> 1) && jj < ii*2 ;jj+=ii/4);
            FFTLEN = jj;
            g_fftlen = jj;
        }
        if(s_FFTLEN > 0)
            {
                FFTLEN = s_FFTLEN;
                g_fftlen = s_FFTLEN;
                s_FFTLEN = 0;
            }
        if (FFTLEN < STRIDE_DIM*2)
            FFTLEN = fftdwt;
	avg_num_b_per_word = ((zpad || generic) ? ng * 2.0 : (logb(k) + ng)) / FFTLEN;
    }   // End special IBDWT code.
    
    tbpw = (zpad || generic) ? 2.0*bpw + bitsmc : 2.0*bpw + kbits + bitsmc + 0.6*log((double)FFTLEN)/log(2.0);
    if (debug && !zpad && !generic) {
        sprintf (gwypbuf, "FFTLEN = %d, bpw = %f, Bits per double = %f, Maxbpd = %f\n",
        FFTLEN, bpw, tbpw, MAXBITSPERDOUBLE);
        if (printfunction != NULL)
            (*printfunction)(gwypbuf);
    }
    return (zpad);
}

void gwypset_larger_fftlen_count(
    int count
)
{
    FFTINC = count;
}

void	init_fftw (int n) {
    int j, hn = n/2;

    if (compl2) {   // Complex DWT
        cnp = (double*)malloc(2*FFTLEN*sizeof(double));  // *2
        nb_malloc++;
        snp = (double*)malloc(2*FFTLEN*sizeof(double));  // *2
        nb_malloc++;
        for (j=0;j<FFTLEN;j++)  {
            cnp[j] = fftcosp((unsigned long)j, n);
            snp[j] = fftsinp((unsigned long)j, n);
        }
        cxin = (fftw_complex *) malloc (n * sizeof(fftw_complex));  // *2
        nb_malloc++;
        cyin = (fftw_complex *) malloc (n * sizeof(fftw_complex));  // *2
        nb_malloc++;
        cxout = (fftw_complex *) malloc (n * sizeof(fftw_complex));  // *2
        nb_malloc++;
        cyout = (fftw_complex *) malloc (n * sizeof(fftw_complex));  // *2
        nb_malloc++;
        cutilSafeCall(hipMalloc((void**)&cuda_cxin, 
                sizeof(fftw_complex)*(n+STRIDE_DIM*STRIDE_DIM)));  // *2
        nb_cudamalloc++;
        cutilSafeCall(hipMalloc((void**)&cuda_cyin, 
                sizeof(fftw_complex)*(n+STRIDE_DIM*STRIDE_DIM)));  // *2
            nb_cudamalloc++;
        cutilSafeCall(hipMalloc((void**)&cuda_cxout, 
                sizeof(fftw_complex)*(n+STRIDE_DIM*STRIDE_DIM)));  // *2
        nb_cudamalloc++;
        cutilSafeCall(hipMalloc((void**)&cuda_cyout, 
                sizeof(fftw_complex)*(n+STRIDE_DIM*STRIDE_DIM)));  // *2
            nb_cudamalloc++;
        cufftSafeCall(hipfftPlan1d(&cuda_fwpx,FFTLEN/2, HIPFFT_Z2Z, 1));
        if (!cufftonly) {
            cutilSafeCall(hipMalloc((void**)&cuda_cnp, 
                2*sizeof(double)*(n+STRIDE_DIM*STRIDE_DIM)));  // *2
            nb_cudamalloc++;
            cutilSafeCall(hipMalloc((void**)&cuda_snp,
                2*sizeof(double)*(n+STRIDE_DIM*STRIDE_DIM)));  // *2
            nb_cudamalloc++;
            cutilSafeCall(hipMemcpy(cuda_cnp,cnp,n*sizeof(double),hipMemcpyHostToDevice));
            cutilSafeCall(hipMemcpy(cuda_snp,snp,n*sizeof(double),hipMemcpyHostToDevice));
            cutilSafeCall(hipMalloc((void**)&cuda_x, 
                sizeof(fftw_complex)*(n+STRIDE_DIM*STRIDE_DIM)));  // *2
            nb_cudamalloc++;
            cutilSafeCall(hipMalloc((void**)&cuda_y, 
                sizeof(fftw_complex)*(n+STRIDE_DIM*STRIDE_DIM)));  // *2
            nb_cudamalloc++;
            if (generic) {
                cutilSafeCall(hipMalloc((void**)&cuda_tmp_g, 
                    sizeof(fftw_complex)*(n+STRIDE_DIM*STRIDE_DIM)));  // *2
                nb_cudamalloc++;
                cutilSafeCall(hipMalloc((void**)&cuda_m, 
                    sizeof(fftw_complex)*(n+STRIDE_DIM*STRIDE_DIM)));  // *2
                nb_cudamalloc++;
                cutilSafeCall(hipMemcpy(cuda_m,modulus,sizeof(double)*n,hipMemcpyHostToDevice));
                cutilSafeCall(hipMalloc((void**)&cuda_r, 
                    sizeof(fftw_complex)*(n+STRIDE_DIM*STRIDE_DIM)));  // *2
                nb_cudamalloc++;
                cutilSafeCall(hipMemcpy(cuda_r,recip,sizeof(double)*n,hipMemcpyHostToDevice));
                cutilSafeCall(hipMalloc((void**)&cuda_cm, 
                    sizeof(fftw_complex)*(n+STRIDE_DIM*STRIDE_DIM)));  // *2
                nb_cudamalloc++;
                cuda_fold_kernel<<<(hn+127)/128,128>>>(cuda_m,cuda_cxin,hn);
                cuda_cnp_m_snp_kernel<<<(hn+127)/128,128>>>(cuda_cxin,cuda_cnp,cuda_snp, hn);
                cufftSafeCall(hipfftExecZ2Z(cuda_fwpx,(hipfftDoubleComplex *)cuda_cxin,
                    (hipfftDoubleComplex *)cuda_cm,HIPFFT_FORWARD));
                cutilSafeCall(hipMalloc((void**)&cuda_cr, 
                    sizeof(fftw_complex)*(n+STRIDE_DIM*STRIDE_DIM)));  // *2
                nb_cudamalloc++;
                cuda_fold_kernel<<<(hn+127)/128,128>>>(cuda_r,cuda_cxin,hn);
                cuda_cnp_m_snp_kernel<<<(hn+127)/128,128>>>(cuda_cxin,cuda_cnp,cuda_snp, hn);
                cufftSafeCall(hipfftExecZ2Z(cuda_fwpx,(hipfftDoubleComplex *)cuda_cxin,
                    (hipfftDoubleComplex *)cuda_cr,HIPFFT_FORWARD));
            }   // End generic
        }       // End !cufftonly
    }           // End Complex DWT
    else {      // Real DWT
        xin = (double *) malloc (n * sizeof(double));
        nb_malloc++;
        yin = (double *) malloc (n * sizeof(double));
        nb_malloc++;
        cxout = (fftw_complex *) malloc (2*(hn+1) * sizeof(fftw_complex));  // *2
        nb_malloc++;
        cyout = (fftw_complex *) malloc (2*(hn+1) * sizeof(fftw_complex));  // *2
        nb_malloc++;
        cutilSafeCall(hipMalloc((void**)&cuda_xin, sizeof(double)*(n+STRIDE_DIM*STRIDE_DIM)));
        nb_cudamalloc++;
        cutilSafeCall(hipMalloc((void**)&cuda_yin, sizeof(double)*(n+STRIDE_DIM*STRIDE_DIM)));
        nb_cudamalloc++;
        cutilSafeCall(hipMalloc((void**)&cuda_cxout, sizeof(double)*2*(hn+1)));
        nb_cudamalloc++;
        cutilSafeCall(hipMalloc((void**)&cuda_cyout, sizeof(double)*2*(hn+1)));
        nb_cudamalloc++;
        cufftSafeCall(hipfftPlan1d(&cuda_fwpx,FFTLEN, HIPFFT_D2Z, 1));		cufftSafeCall(hipfftPlan1d(&cuda_bwpx,FFTLEN, HIPFFT_Z2D, 1));
        if (!cufftonly) {
            cutilSafeCall(hipMalloc((void**)&cuda_x, sizeof(double)*(n+STRIDE_DIM*STRIDE_DIM)));
            nb_cudamalloc++;
            cutilSafeCall(hipMalloc((void**)&cuda_y, sizeof(double)*(n+STRIDE_DIM*STRIDE_DIM)));
            nb_cudamalloc++;
            cutilSafeCall(hipMalloc((void**)&cuda_cxin, 
                    sizeof(fftw_complex)*(n+STRIDE_DIM*STRIDE_DIM)));  // *2
            nb_cudamalloc++;
            cutilSafeCall(hipMalloc((void**)&cuda_cyin, 
                    sizeof(fftw_complex)*(n+STRIDE_DIM*STRIDE_DIM)));  // *2
            nb_cudamalloc++;
            if (generic) {
                cutilSafeCall(hipMalloc((void**)&cuda_tmp_g, 
                    2*sizeof(double)*(n+STRIDE_DIM*STRIDE_DIM)));  // *2
                nb_cudamalloc++;
                cutilSafeCall(hipMalloc((void**)&cuda_m, 
                    2*sizeof(double)*(n+STRIDE_DIM*STRIDE_DIM)));  // *2
                nb_cudamalloc++;
                cutilSafeCall(hipMemcpy(cuda_m,modulus,sizeof(double)*n,hipMemcpyHostToDevice));
                cutilSafeCall(hipMalloc((void**)&cuda_r, 
                    2*sizeof(double)*(n+STRIDE_DIM*STRIDE_DIM)));  // *2
                nb_cudamalloc++;
                cutilSafeCall(hipMemcpy(cuda_r,recip,sizeof(double)*n,hipMemcpyHostToDevice));
                cutilSafeCall(hipMalloc((void**)&cuda_cm, 
                    2*sizeof(double)*2*(hn+1)));  // *2
                nb_cudamalloc++;
                cufftSafeCall(hipfftExecD2Z(cuda_fwpx,(hipfftDoubleReal *)cuda_m,(hipfftDoubleComplex
                    *)cuda_cm));
                cutilSafeCall(hipMalloc((void**)&cuda_cr, 
                    2*sizeof(double)*2*(hn+1)));  // *2
                nb_cudamalloc++;
                cufftSafeCall(hipfftExecD2Z(cuda_fwpx,(hipfftDoubleReal *)cuda_r,
                    (hipfftDoubleComplex *)cuda_cr));
            }   // End generic
        }       // End !cufftonly
    }           // End Real DWT
}

// Get the pointers to the user output functions

void gwypsetoutputs (
    void(*screenf)(char *),
    void(*bothf)(char *)
)
{
    screen_output = screenf;
    both_output = bothf;
}

// Initialize the gwypnum system

int
gwypsetup(
    double          k,	// The multiplier
    unsigned long   b,	// The base (force generic reduction if not two)
    unsigned long   n,	// The exponent
    signed long     c,	// c, in k*b^n+c
    giant           modulus_arg	// The modulus of the modular reduction
)
{
    long    j, len/*, g /*limit*/;
    double  dlimit, min_hlimit = 0.0, max_hlimit = 0.0;
    double  log2k, log2b;
    double  tc1 = 12345.6789, tc2 = 6789.12345;

    nb_malloc = 0, nb_free = 0, nb_cudamalloc = 0, nb_cudafree = 0; // 03/02/21
    if(bitlen (modulus_arg) < 128*8)    {
        return(1);  // Make only an APRCL test...
    } 	
    printfunction = (verbose)? both_output : screen_output;
    dd_data_arg = gwdbldbl_data_alloc ();
    gmodulus = modulus_arg;    
//    if (abs(c) != 1)
//        generic = 1;
    log2b = log2(b);
    if (k > 0.0) {
        log2k = log2(k);
        log2ofkbpown = (double)n*log2b + log2k;
    }
    else
        log2ofkbpown = (double)n*log2b;
    if (b != 2 || k == 0.0 || k*MAXMULCONST > MAX_ZEROPAD_K)
        generic = 1;
    MAXERR = 0.0;
    BIGVAL = 3.0;
    while (RINTP(tc1) != RINT(tc1) || RINTP(tc2) != RINT(tc2))
        BIGVAL *= 2.0;
    while (RINTP(tc1) == RINT(tc1) && RINTP(tc2) == RINT(tc2))
        BIGVAL *= 2.0;
    BIGVAL /= 2.0;
    bit_length = bitlen (gmodulus);
    if (debug) {
        sprintf (gwypbuf, "k = %14.1f, b = %lu, n = %lu, c = %ld, log2ofkbpown = %14.5f, bit_length = %lu\n", k, b, n, c, log2ofkbpown, bit_length);
        if (printfunction != NULL)
            (*printfunction)(gwypbuf);
    }
    kg = k;
    bg = b;
    ng = n;
    cg = c;
    cn=sn=cnp=snp=two_to_phi=two_to_minusphi=NULL;
    invlimit=flimit=hlimit=limitbv=scr=scral=NULL;
    fftbase=NULL;
    modulus=recip=NULL;
    grecip=NULL;
    GWP_RANDOM=NULL;
    gwyptmp=NULL;   // JP 27/11/18
    zp = set_fftlen(k, b, n, c);
    if(FFTLEN < 128 && ((k!=1) || (b!=4) || (c!=1))) {
        return(1);  // Make only an APRCL test...
    }
    k = kg;
    b = bg;
    n = ng;
    c = cg;
    compl2 = (c >= 1)? 1 : 0;   // 23/03/21
    plus = compl2;
    NUM_B_PER_SMALL_WORD = (unsigned long) avg_num_b_per_word;
    if (generic) {
        bits = ng/FFTLEN;
        limit_high = (double)(1<<bits);
        limit_inverse_high = 1.0/limit_high;
        limit_high_bigval = (limit_high*BIGVAL2)-BIGVAL2;
    }
    gwfft_weight_setup (dd_data_arg, zp, k, b, n, c, FFTLEN);
    len = 2*(FFTLEN+16)*sizeof(double);
    fftbase = (int *)malloc((FFTLEN+16)*sizeof(int));
    nb_malloc++;
    if (!generic && !zp) {
        two_to_phi = (double *)malloc(len);
        nb_malloc++;
        two_to_minusphi = (double *)malloc(len);
        nb_malloc++;
    }
    
    invlimit = (double *)malloc(len);
    nb_malloc++;
    flimit = (double *)malloc(len);
    nb_malloc++;
    hlimit = (double *)malloc(len);
    nb_malloc++;
    limitbv = (double *)malloc(len);
    nb_malloc++;
    if (!cufftonly) {
        if (!generic && !zp) {
            cutilSafeCall(hipMalloc((void**)&cuda_two_to_phi, len
                +STRIDE_DIM*STRIDE_DIM*sizeof(double)));
            nb_cudamalloc++;
            cutilSafeCall(hipMalloc((void**)&cuda_two_to_minusphi, len
                +STRIDE_DIM*STRIDE_DIM*sizeof(double)));
            nb_cudamalloc++;
        }
        cutilSafeCall(hipMalloc((void**)&g_invlimit, len+STRIDE_DIM*STRIDE_DIM*sizeof(double)));
        nb_cudamalloc++;
        cutilSafeCall(hipMalloc((void**)&g_hlimit, len+STRIDE_DIM*STRIDE_DIM*sizeof(double)));
        nb_cudamalloc++;
        cutilSafeCall(hipMalloc((void**)&g_limitbv, len+STRIDE_DIM*STRIDE_DIM*sizeof(double)));
        nb_cudamalloc++;
        cutilSafeCall(hipMalloc((void**)&cuda_tmp, len+STRIDE_DIM*STRIDE_DIM*sizeof(double)));
        nb_cudamalloc++;
        cutilSafeCall(hipMalloc((void**)&g_carry,len));
        nb_cudamalloc++;
        cutilSafeCall(hipMalloc((void**)&g_err,2*sizeof(float)*(FFTLEN+16)));
        nb_cudamalloc++;
    }
    l_err = (float *)malloc(2*sizeof(float)*(FFTLEN+16));
    nb_malloc++;
    high = (double)intpow(b, gwfft_base(dd_data_arg, 1));
    low = 0.5*high;
    last = (double)intpow(b, (gwfft_base(dd_data_arg, FFTLEN)-gwfft_base(dd_data_arg, FFTLEN-1)));
    highinv = 1.0/high;
    lowinv = 1.0/low;
    lastinv = 1.0/last;        
    if (debug && !zp && !generic) {
        sprintf (gwypbuf, "clog2k = %d, log2k = %7.4f, high = %7.4f, low = %7.4f, last = %7.4f\n", (int)ceil(log2k), log2k, high, low, last);
        if (printfunction != NULL)
            (*printfunction)(gwypbuf);
    }	
    wrapindex = 0;
    wrapfactor = 1.0;
//    wrapfactor = (double)abs(c); // 18/03/21
    if (k != 1.0) {
        wrapfactor = (double)intpow((double)b, (double)ceil(logb(k)))-k;
//    if (k != 1.0 || abs(c) != 1)    {
        while (n > gwfft_base(dd_data_arg, wrapindex))
            wrapindex++;
        wrapindex--;
        for (j=0;j+gwfft_base(dd_data_arg, wrapindex)<n;j++)
//            wrapfactor *= (double)b;
            wrapfactor = wrapfactor * (double)b;  // 03/04/21 RINT ??
    }	
/*        sprintf (gwypbuf, "wrapindex = %d, wrapfactor = %f\n", wrapindex, wrapfactor);
        if (printfunction != NULL)
            (*printfunction)(gwypbuf); */
    if (debug && !zp && !generic) {
        sprintf (gwypbuf, "wrapindex = %d, wrapfactor = %f\n", wrapindex, wrapfactor);
        if (printfunction != NULL)
            (*printfunction)(gwypbuf);
        sprintf(gwypbuf, "INIT : log2ofkbpown = %7.4f, low = %7.4f, high = %7.4f, last = %7.4f, B = %g\n", log2ofkbpown, low, high, last, BIGVAL2); // JP 08/07/17
        if (printfunction != NULL)
            (*printfunction)(gwypbuf);
    }	
    maxbitsinfftlen = 0;        // JP 19/09/20
    maxbitsinfftword =0;        // JP 20/09/20
    for(j=0; j<FFTLEN; ++j) {
        if (!generic && !zp) {
            two_to_phi[j] = gwfft_weight (dd_data_arg, j);
            two_to_minusphi[j] = (compl2) ? 2.0*gwfft_weight_inverse_over_fftlen (dd_data_arg, j) :
            gwfft_weight_inverse_over_fftlen (dd_data_arg, j);
        }
        fftbase[j] = gwfft_base(dd_data_arg, j+1) - gwfft_base(dd_data_arg, j);
        maxbitsinfftlen += abs(fftbase[j]); // JP 19/09/20 , 04/04/21
        if (maxbitsinfftword < fftbase[j])
            maxbitsinfftword = fftbase[j];  // JP 20/09/20
    }
    if (!cufftonly) {
        if (!generic && !zp) {
            cutilSafeCall(hipMemcpy(cuda_two_to_phi,two_to_phi,FFTLEN*sizeof(double),hipMemcpyHostToDevice));
            cutilSafeCall(hipMemcpy(cuda_two_to_minusphi,two_to_minusphi,FFTLEN*sizeof(double),hipMemcpyHostToDevice));
        }
        for(j=0; j<FFTLEN; ++j)
            l_err[j]=0;
        cutilSafeCall(hipMemcpy(g_err,l_err,FFTLEN*sizeof(float),hipMemcpyHostToDevice));
    }
    ttmp = (compl2) ?
            2.0*gwfft_weight_inverse_over_fftlen
            (dd_data_arg, 0) : gwfft_weight_inverse_over_fftlen (dd_data_arg, 0);
    flimit[0] = high;
    invlimit[0] = highinv;
    hlimit[0] = low;
    limitbv[0] = high*BIGVAL2-BIGVAL2;
    flimit[FFTLEN-1] = last;
    invlimit[FFTLEN-1] = lastinv;
    hlimit[FFTLEN-1] = 0.5*last;
    limitbv[FFTLEN-1] = last*BIGVAL2-BIGVAL2;
        
    min_hlimit = min (low, (0.5*last));
    max_hlimit = max (low, (0.5*last));
    
    for(j=1; j<FFTLEN-1; ++j) {
        dlimit = (double)intpow((double)b, (double)(gwfft_base(dd_data_arg, j+1) - gwfft_base(dd_data_arg, j)));
        flimit[j] = dlimit;
        invlimit[j] = 1.0/dlimit;
        hlimit[j] = 0.5*dlimit;
        limitbv[j] = dlimit*BIGVAL2-BIGVAL2;
        if (min_hlimit > hlimit[j])
            min_hlimit = hlimit[j];
        if (max_hlimit < hlimit[j])
            max_hlimit = hlimit[j];
    }
    if (min_hlimit > hlimit[FFTLEN-1])
        min_hlimit = hlimit[FFTLEN-1];
    if (max_hlimit < hlimit[FFTLEN-1])
        max_hlimit = hlimit[FFTLEN-1];

    dim3 grid(STRIDE_DIM/BLOCK_DIM,STRIDE_DIM/BLOCK_DIM, 1);
    dim3 threads(BLOCK_DIM, BLOCK_DIM, 1);

    if (!cufftonly) {
        cutilSafeCall(hipMemcpy(cuda_tmp,invlimit,FFTLEN*sizeof(double),hipMemcpyHostToDevice));
        for(j=0;j<FFTLEN;j+=(STRIDE_DIM*STRIDE_DIM))
           	transpose<<<grid, threads>>>((double *)&g_invlimit[j],(double *)&cuda_tmp[j],(int) STRIDE_DIM,(int) STRIDE_DIM);
        cutilSafeCall(hipMemcpy(cuda_tmp,limitbv,FFTLEN*sizeof(double),hipMemcpyHostToDevice));
        for(j=0;j<FFTLEN;j+=(STRIDE_DIM*STRIDE_DIM))
           	transpose<<<grid, threads>>>((double *)&g_limitbv[j],(double *)&cuda_tmp[j],(int) STRIDE_DIM,(int) STRIDE_DIM);
        cutilSafeCall(hipMemcpy(g_hlimit,hlimit,FFTLEN*sizeof(double),hipMemcpyHostToDevice));
    }
    init_fftw((FFTLEN < STRIDE_DIM*2)?STRIDE_DIM*2:FFTLEN);
    addinindex = 0;
    addinvalue = 0.0;
    SMALLMULCONST = 1.0;
    MULBYCONST = 0;
    setup = TRUE;
    return 0;
}

int gwypsetup_general_mod_giant (
    giant modulus_arg	// The modulus of the modular reduction
)
{
    generic = TRUE;
//    return gwypsetup (0.0, 1, 1, -1, modulus_arg);
    return gwypsetup (0.0, 2, 1, -1, modulus_arg);
}

//	Miscellanous utility routines

/* Some words in the FFT data contain floor(p/N), some words contain  floor(p/N)+1 bits.
This function returns TRUE in the latter case. */

int is_big_word (
    unsigned long i)
{
    unsigned long base, next_base;

/* Compute the number of b in this word.
It is a big word if the number of b is more than NUM_B_PER_SMALL_WORD. */

    base = gwfft_base (dd_data_arg, i);
    next_base = gwfft_base (dd_data_arg, i+1);
    return ((next_base - base) > NUM_B_PER_SMALL_WORD);
}

/* Routine map a "bit" number into an FFT word and a "bit" within that word */
/* If b != 2, this routine locates the nth b amongst the FFT words. */

void bitaddr (
    unsigned long bit,
    unsigned long *word,
    unsigned long *bit_in_word,
    unsigned long FFTLEN)
{

/* What word is the bit in? */

    *word = (unsigned long) ((double) bit / avg_num_b_per_word);
    if (*word >= FFTLEN) *word = FFTLEN - 1;

/* Compute the bit within the word. */

    *bit_in_word = bit - gwfft_base (dd_data_arg, *word);
}

/* Routine that tells the assembly code to add a small value to the */
/* results of each multiply. */

void setaddin (
    long    value,
    int     N
)
{
    unsigned long word = 0, b_in_word = 0;  // must be defaulted to zero! 25/03/21

//	ASSERTG (k == 1.0 || abs (c) == 1);

/* In a zero-padded FFT, the value is added into ZPAD0 */
        
    if (zp || generic) {
        addinvalue = (double) value;
        addinindex = 0;
        return;
    }

/* If value is a multiple of b, "shift" it right and increment b count.  This will ensure that we modify the proper FFT word. */

    for (b_in_word = 0; value && ((value % bg) == 0); value /= (long)bg) // Take care to integer type!!
        b_in_word++;
        
/* Convert the input value to 1/k format.
Case 1 (k*b^n-1): Inverse of k is b^n.
Case 3 (k*b^n+1): Inverse of k is -b^n.
No other cases can be handled. */

    if (cg == -1) {
        bitaddr (ng + b_in_word, &word, &b_in_word, N);
    }
    else if (cg == 1) {
        bitaddr (ng + b_in_word, &word, &b_in_word, N);
        value = -value;
    }

/* Tell assembly code to add the shifted value to the multiplication result. */

    addinindex = word;
    addinvalue = (double)value * intpow ((double)bg, (double)b_in_word);
}


/* Add a small constant at the specified power of b after the next multiplication.
That is, value*b^power_of_b is added to the next multiplication result.  This only works if k=1. */

void gwypsetaddinatpowerofb (
    long	value,
    unsigned long power_of_b_arg)
{
    unsigned long word, b_in_word, power_of_b;

/* If value is even, shift it right and increment bit number.  This will ensure that we modify the proper FFT word. */

    for (power_of_b = power_of_b_arg; value && (value & 1) == 0; value >>= 1)
        power_of_b++;

    bitaddr (power_of_b, &word, &b_in_word, FFTLEN);

    addinvalue = (double)(value<<b_in_word);
    addinindex = word;
}

/* Test if a gwypnum is zero */

#define	MAX_NZ_COUNT 10

int
gwypiszero(
    gwypnum 	gg
)
{
    register long   j, count = 0;
    register double *gp = gg;
    long            result;
    giant           gtest;

    if (!generic)
        for(j=0; j<FFTLEN; j++) {
            if (count > MAX_NZ_COUNT)
                return 0;
            // Too much non zero words, the gwypnum is not zero.
            else if (*gp++)
                count++;
            // Update the count of non-zero words.
        }
    if (count || generic) {
        // The gwypnum needs a more accurate test...
        gtest = newgiant (2*FFTLEN*sizeof(double)/sizeof(short) + 16);
        // Allocate memory for the giant
        gwyptogiant (gg, gtest);
        result = isZero (gtest);
        gwypfree (gtest);   // Free memory
        return (result);
    }
    else
        return 1;	// The gwypnum is zero
}

/* Test two gwypnums for equality */

int gwypequal (
    gwypnum gw1, 
    gwypnum gw2
) 
{
    gwypnum gwdiff;
    int     result;

    gwdiff = gwypalloc ();
    // Reserve memory for the difference
    gwypsub3 (gw1, gw2, gwdiff);
    // Normalized subtract...
    result = gwypiszero (gwdiff);
    // Test for zero difference
    gwypfree (gwdiff);	// Free memory
    return (result);
}

//	User side functions...

double
gwypnormalize(
    gwypnum s
)
{
    double err;

    if (tdebug)
        gwypstart_timer (2);
    err = (zp || generic)? rnormalize(s, FFTLEN, E_CHK, 0, 0) : inormalize(s, FFTLEN, E_CHK, 0, 0);
    if (tdebug)
        gwypend_timer (2);
    return (err);
}


double
gwyprawnormalize(
    gwypnum s
)
{
    double err;

    if (tdebug)
        gwypstart_timer (2);
    err = (zp || generic)? rnormalize(s, FFTLEN, E_CHK, 1, 1) : inormalize(s, FFTLEN, E_CHK, 1, 1);
    if (tdebug)
        gwypend_timer (2);
    return (err);
}

void cuda_generic_modred(
    gwypnum s,
    int     flag
)
{
    double err;

    gwypcopyzero (s, gwyptmp, zerowordslow);
    err = cuda_lucas_mul (recip, gwyptmp, FFTLEN, E_CHK, 1, 1, flag);
    if (err > MAXERR)
        MAXERR = err;
    gwypsetzero (gwyptmp, zerowordshigh);
    err = cuda_lucas_mul (modulus, gwyptmp, FFTLEN, E_CHK, 1, 1, flag);
    if (err > MAXERR)
        MAXERR = err;
    if (tdebug)
        gwypstart_timer (4);
    if (compl2) {
        gwypaddquick (gwyptmp, s);
        gwypsubquick (modulus, s);
    }
    else
        gwypsubquick (gwyptmp, s);
    if (tdebug)
        gwypend_timer (4);
}

void generic_modred(
    gwypnum s
)
{
    double err;

    gwypcopyzero (s, gwyptmp, zerowordslow);
    err = lucas_mul (recip, gwyptmp, FFTLEN, E_CHK, 1, 1);
    if (err > MAXERR)
        MAXERR = err;
    gwypsetzero (gwyptmp, zerowordshigh);
    err = lucas_mul (modulus, gwyptmp, FFTLEN, E_CHK, 1, 1);
    if (err > MAXERR)
        MAXERR = err;
    if (tdebug)
        gwypstart_timer (4);
    if (compl2) {
        gwypaddquick (gwyptmp, s);
        gwypsubquick (modulus, s);
    }
    else
        gwypsubquick (gwyptmp, s);
    if (tdebug)
        gwypend_timer (4);
}

void gwypfft_description (
    char *buf)
{
    if (zp)
        if (compl2)
            sprintf (buf, "Using complex zero-padded rational base DWT, FFT length = %d", FFTLEN);
        else
            sprintf (buf, "Using real zero-padded rational base DWT, FFT length = %d", FFTLEN);
    else if (generic)
        if (compl2)
            sprintf (buf, "Using complex rational base DWT and generic reduction, FFT length = %d", FFTLEN);
        else
            sprintf (buf, "Using real rational base DWT and generic reduction, FFT length = %d", FFTLEN);
    else    // IBDWT
        if (compl2)
            sprintf (buf, "Using complex irrational base DWT, FFT length = %d", FFTLEN);
        else
            sprintf (buf, "Using real irrational base DWT, FFT length = %d", FFTLEN);
}



// User side large integers arithmetic operations


void gwypadd (
    gwypnum s,
    gwypnum d)
{
    int i;

    for (i=0; i<FFTLEN; i++)
        d[i] += s[i];
    gwyprawnormalize (d);
}


void gwypsub (
    gwypnum s,
    gwypnum d)
{
    int i;

    for (i=0; i<FFTLEN; i++)
        d[i] -= s[i];
    gwyprawnormalize (d);
}

void gwypadd3 (
    gwypnum s1,
    gwypnum s2,
    gwypnum d)
{
    int i;

    for (i=0; i<FFTLEN; i++)
        d[i] = s1[i] + s2[i];
    gwyprawnormalize (d);
}


void gwypsub3 (
    gwypnum s1,
    gwypnum s2,
    gwypnum d)
{
    int i;

    for (i=0; i<FFTLEN; i++)
        d[i] = s1[i] - s2[i];
    gwyprawnormalize (d);
}

void
cuda_gwypsquare (
    gwypnum s,
    int     flag)
{
    double  err;
    
    if (generic)
        err = cuda_lucas_square_generic (s, FFTLEN, E_CHK, 0, 0, flag);
    else
        err = cuda_lucas_square (s, FFTLEN, E_CHK, 0, 0, flag);
    if (err > MAXERR)
        MAXERR = err;
}

void
gwypsquare (gwypnum s)
{
    double err;

    err = lucas_square (s, FFTLEN, E_CHK, 0, 0);
    if (err > MAXERR)
        MAXERR = err;
    if (generic)
        generic_modred (s);
}

void
cuda_gwypmul (
    gwypnum s,
    gwypnum d,
    int     flag
)
{
    double err;
    
    if (generic)
        err = cuda_lucas_mul_generic (s, d, FFTLEN, E_CHK, 0, 0, flag);
    else
        err = cuda_lucas_mul (s, d, FFTLEN, E_CHK, 0, 0, flag);
    if (err > MAXERR)
        MAXERR = err;
}

void
gwypmul (
    gwypnum s,
    gwypnum d
    
)
{
    double  err;

    err = lucas_mul (s, d, FFTLEN, E_CHK, 0, 0);
    if (err > MAXERR)
        MAXERR = err;
    if (generic)
        generic_modred (d);
}

/* Generate random FFT data */

void gwyp_random_number
(
    gwypnum x
)
{
    giant           g;
    unsigned long   i, len;

/* Generate the random number */

    srand ((unsigned int) time (NULL));
    len = (unsigned long) (FFTLEN*sizeof(double)/sizeof(short));
    g = newgiant (2*len + 16);
    for (i = 0; i < len; i++) {
        g->n[i] = (unsigned short) rand();
    }
    g->sign = len;
    modg (gmodulus, g);
    gianttogwyp (g, x);
    gwyprawnormalize (x);
    gwypfree(g);
}


/* Square a number using a slower method that will have reduced round-off error on non-random input data.*/

void gwypsquare_carefully
(
    gwypnum s  /* Source and destination */
)
{
    gwypnum tmp1, tmp2;
    double  err;

/* Generate a random number, if we have't already done so */
    if (GWP_RANDOM == NULL) {
        GWP_RANDOM = gwypalloc ();
        gwyp_random_number (GWP_RANDOM);
    }

/* Now do the squaring using three multiplies and adds */

    tmp1 = gwypalloc ();
    tmp2 = gwypalloc ();
    gwypadd3 (s, GWP_RANDOM, tmp1);
    /* Compute s+random */
    gwypcopy (GWP_RANDOM, tmp2);
    err = lucas_mul (tmp2, s, FFTLEN, E_CHK, 1, 0);	/* Compute s*random without addin*/
    if (err > MAXERR)
        MAXERR = err;
    if (generic)
        generic_modred (s);
    err = lucas_square (tmp2, FFTLEN, E_CHK, 1, 0);	/* Compute random^2 without addin*/
    if (err > MAXERR)
            MAXERR = err;
    if (generic)
            generic_modred (tmp2);
    gwypsquare (tmp1);					/* Compute (s+random)^2  + addinvalue */
    gwypsubquick (tmp2, tmp1);			/* Calc s^2 from 3 results */
    gwypaddquick (s, s);
    gwypsub3 (tmp1, s, s);

/* Free memory and return */

    gwypfree (tmp1);
    gwypfree (tmp2);
}


/* Multiply numbers using a slower method that will have reduced */
/* round-off error on non-random input data.*/

void gwypmul_carefully
(
    gwypnum s,
    gwypnum t   /* Source and destination */
)
{
    gwypnum tmp1, tmp2, tmp3, tmp4;
    double  err;

/* Generate a random number, if we have't already done so */

    if (GWP_RANDOM == NULL) {
        GWP_RANDOM = gwypalloc ();
        gwyp_random_number (GWP_RANDOM);
    }

/* Now do the multiply using four multiplies and adds */

    tmp1 = gwypalloc ();
    tmp2 = gwypalloc ();
    tmp3 = gwypalloc ();
    tmp4 = gwypalloc ();
    gwypcopy (s, tmp4);
    gwypadd3 (s, GWP_RANDOM, tmp1);
        /* Compute s+random */
    gwypadd3 (t, GWP_RANDOM, tmp3);
        /* Compute t+random */
    gwypcopy (GWP_RANDOM, tmp2);
    err = lucas_mul (tmp2, tmp4, FFTLEN, E_CHK, 1, 0); /* Compute s*random without adding*/
    if (err > MAXERR)
        MAXERR = err;
    if (generic)
        generic_modred (tmp4);
    err = lucas_mul (tmp2, t, FFTLEN, E_CHK, 1, 0);	    /* Compute t*random without adding*/
    if (err > MAXERR)
        MAXERR = err;
    if (generic)
        generic_modred (t);
    err = lucas_square (tmp2, FFTLEN, E_CHK, 1, 0);	    /* Compute random^2 without addin*/
    if (err > MAXERR)
        MAXERR = err;
    if (generic)
        generic_modred (tmp2);
    err = lucas_mul (tmp1, tmp3, FFTLEN, E_CHK, 0, 0);
    /* Compute (s+random)*(t+random) + addinvalue */
    if (err > MAXERR)
        MAXERR = err;
    if (generic)
        generic_modred (tmp3);
    gwypsubquick (tmp2, tmp3);
        /* Subtract random^2 */
    gwypsubquick (t, tmp3);
    gwypsub3 (tmp3, tmp4, t);

/* Free memory and return */

    gwypfree (tmp1);
    gwypfree (tmp2);
    gwypfree (tmp3);
    gwypfree (tmp4);
}

// Set small add-in constant

void gwypsetaddin(
    long s
)
{
    setaddin (s, FFTLEN);
}

// Set small multiplicative constant

void gwypsetmulbyconst(
    long s
)
{
    SMALLMULCONST = (double)s;
}

// Set the maximum of the multiplicative constant

void gwypsetmaxmulbyconst(
    long s
)
{
    MAXMULCONST = (double)s;
}

// Conversion routines

void itogwyp(	// Set a gwypnum to a small value
    int     s,
    gwypnum d
)
{
    int     j, saveindex;
    double  savevalue;
    if (!generic && !zp && (kg != 1.0) && (abs(cg) != 1)) {  // 28/02/21
        giant gs = newgiant (4);
        itog (s, gs);
        gianttogwyp (gs, d);
        gwypfree (gs);
        return;
    }
    for (j=0; j<FFTLEN; j++)
    // Init the large integer to zero
        d[j] = 0.0;
    saveindex = addinindex;
    // Save setaddin internal status
    savevalue = addinvalue;
    gwypsetaddin(s);
    d[addinindex] = addinvalue;
    // Set the large integer value in 1/k format
    addinindex = saveindex; // Restore setaddin
    addinvalue = savevalue;
}

void gwypaddsmall( // Add a small value to a gwypnum
    gwypnum d,
    int     s
)
{
    int     saveindex;
    double  savevalue;

    saveindex = addinindex;
    // Save setaddin internal status
    savevalue = addinvalue;
    gwypsetaddin (s);
    gwypnormalize(d);
    // Normalize the number while adding the value
    addinindex = saveindex; // Restore setaddin
    addinvalue = savevalue;
}

/* Internal recursive routine to convert a giant to gwnum FFT format. */

long nonbase2_gianttogw (
    giant           a,
    gwypnum	    g,
    unsigned long   limit,
        /* How many FFT words to set */
    unsigned long   offset,
        /* Offset into FFT array of words to set */
    long            carry)
        /* Carry to add into this section */        
{
    ASSERTG (a->sign >= 0);
        /* We only handle positive numbers */
/* If we are converting a lot of words, divide and conquer. */
    if (limit >= 50) {
        giant           upper, tmp;
        int             num_b;
        unsigned long   half_limit = limit >> 1;
        tmp = popg ();
        upper = popg ();
        num_b = gwfft_base (dd_data_arg, offset + half_limit) - gwfft_base (dd_data_arg, offset);
        powerg (bg, num_b, tmp);
        gtog (a, upper);
        divg (tmp, upper);
        mulg (upper, tmp);
        subg (tmp, a);
        carry = nonbase2_gianttogw (a, g, half_limit, offset, carry);
        carry = nonbase2_gianttogw (upper, g, limit - half_limit, offset + half_limit, carry);
        pushg (2);
    }
/* Convert the giant to FFT format */
    else {
        giant           newg, tmp;
        unsigned long   i, mask1, mask2;
        long            value;
        newg = popg ();
        tmp = popg ();
        mask1 = intpow (bg, NUM_B_PER_SMALL_WORD);
        mask2 = bg * mask1;
        for (i = offset; i < offset + limit; i++) {
            unsigned long mask;
            mask = is_big_word (i) ? mask2 : mask1;
            gtog (a, newg);
            if (i != FFTLEN - 1) {
                itog (mask, tmp);
                divg (tmp, a);
                mulg (a, tmp);
                subg (tmp, newg);
                // newg = a modulo mask
            }
            value = (newg->sign) ? newg->n[0] : 0;
            value += (newg->sign > 1)? newg->n[1]<<16 : 0;
            value += carry;
            if (value > (long) (mask >> 1) && i != FFTLEN - 1) {
                value = value - mask;
                carry = 1;
            }
            else {
                carry = 0;
            }
            g[i] = value;
        }
        pushg (2);
    }
/* Return carry for next section */
    return (carry);
}

/* Convert a giant to gwypnum FFT format.  Giant must be a positive number. */

void gianttogwyp (
    giant   a,
    gwypnum g
)
{
    giant           newg;
    unsigned        e1len;
    int	            i, bits_in_next_binval;
    unsigned long   binval, carry, limit, *input;
    unsigned long   *e1;

/* To make the mod k*b^n+c step faster, gwypnum's are pre-multiplied by 1/k */
/* If k is greater than 1, then we calculate the inverse of k, multiply the giant by the inverse of k, and do a mod k*b^n+c. */

    newg = newgiant (((unsigned long)(bit_length/16) + 1) * 2);

    if (kg != 1.0) {
/* Easy case 1 (k*b^n-1): Inverse of k is b^n */
        if (cg == -1) {
            if (bg == 2) {
                gtog (a, newg);
                gshiftleft (ng, newg);
            }
            else {
                powerg (bg, ng, newg);
                mulg (a, newg);                        
            }
        }
/* Easy case 2 (k*b^n+1): Inverse of k is -b^n */
        else if (cg == 1) {
            gtog (gmodulus, newg);
// make -a a positive number!
            subg (a, newg);
            if (bg == 2)    
                gshiftleft (ng, newg);
            else {
                gtog (newg, a);
                powerg (bg, ng, newg);
                mulg (a, newg);                        
            }
        }
        else {
            gtog (gk, newg);
            invg (gmodulus, newg);
            mulg (a, newg);
        }
        modg (gmodulus, newg);
    }
    else
        gtog (a, newg);
/* Now convert the giant to FFT format */
    ASSERTG (a->sign >= 0);
/* Figure out how many FFT words we will need to set */
    limit = (unsigned long)ceil ((double)bitlen (newg) / (avg_num_b_per_word * log2 (bg)));
    if (limit > FFTLEN) limit = FFTLEN;
// Now convert the giant to FFT format. For base 2 we simply copy bits.  
    if (bg == 2) {
/* FFT words may be larger than 16 bits, so we convert the input
 * data from the giant in a 32 bits array. */
        e1len = newg->sign + 2;
        input = (unsigned long *) malloc (e1len*sizeof (unsigned long));
        e1len = newg->sign/2;
        if (newg->sign & 1)
            e1len++;
        gwypgianttoula (newg, input); // copy in the 32 bits array
        e1 = input;
        if (e1len) {
            binval = *e1++; e1len--; bits_in_next_binval = 32;                
        } // First input word
        else 
            binval = 0;
        carry = 0;
        for (i = 0; i < limit; i++) {
            int	    bits;
            long    value, mask;
            bits = (zp || generic)? ng/FFTLEN : fftbase [i];
                // bits in current word
            mask = (1L << bits) - 1;
                // max value of current word
            if (i == limit - 1)
                value = binval;
                // last output value
            else
                value = binval & mask;
                // current output value
            value = value + carry;
                // add previous carry
            if (value > (mask >> 1) && bits > 1 && i != (FFTLEN - 1)) {
                value = value - (mask + 1);
                // Test if balanced repr. yields a carry
                carry = 1;
            }
            else {
                carry = 0;
            }
            g[i] = (double)value;  // copy the value
            binval >>= bits;
            if (e1len == 0)
                continue;
            if (bits_in_next_binval < bits) {
                if (bits_in_next_binval)
                    binval |= (*e1 >> (32 - bits_in_next_binval)) << (32 - bits);
                bits -= bits_in_next_binval;
                e1++;
                e1len--;
                bits_in_next_binval = 32;
                if (e1len == 0)
                    continue;
            }
            if (bits) {
                binval |= (*e1 >> (32 - bits_in_next_binval)) << (32 - bits);
                bits_in_next_binval -= bits;
            }
        }
        gwypfree (input);
    }
/* Otherwise (non-base 2), we do a recursive divide and conquer radix conversion. */
/* The recursive routine writes on a, so make a copy before calling */
    else {
        if (a != newg) {
            newg = popg ();
            gtog (a, newg);
            a = newg;
        }
        carry = nonbase2_gianttogw (a, g, limit, 0, 0);
    }
/* Write carry, if any, to FFT data */
    if (carry) 
        g[limit++] = carry;
/* Clear the upper words */
    for (i = limit; i < FFTLEN; i++)
        g[i] = 0;
//    gwypnormalize (g);
/* Free allocated memory */
    if (bg !=2)
        pushg (1);
/* Free allocated memory */
//    if (kg > 1.0)
        gwypfree(newg);
}

int gwyptogiant (
    gwypnum gg,
    giant   v)
{
    int retval;
    retval = gwyptogiantnomodulo (gg, v);
    modg (gmodulus, v);
    return (retval);
}


char	timebuf[40];

void	gwypdone (void) {  // Free all the memory used by this code.
    if (!setup)         // Do nothing if gwypsetup has not been called...
        return;
    printfunction = (verbose)? both_output : screen_output;
    MAXMULCONST = 1.0;
    gwypfree (fftbase);
    nb_free++;
    gwypfree (invlimit);
    nb_free++;
    gwypfree (flimit);
    nb_free++;
    gwypfree (hlimit);
    nb_free++;
    gwypfree (limitbv);
    nb_free++;
    gwypfree (l_err);
    nb_free++;
    gwypfree (cxout);   // 27/01/21
    nb_free++;
    gwypfree (cyout);   // 27/01/21
    nb_free++;
    gwypfree (dd_data_arg);
    if (!generic && !zp) {
        gwypfree (two_to_phi);
        nb_free++;
	gwypfree (two_to_minusphi);
        nb_free++;
    }
    if (zp) {
        gwypfree (scr);
        nb_free++;
    }
    if (generic) {
        gwypfree (modulus);
        nb_free++;
	gwypfree (recip);
        nb_free++;
	gwypfree (gwyptmp);
        nb_free++;
        gwypfree (grecip);
        nb_free++;
    }
    if (!cufftonly) {
        if (!generic && !zp) {
            cutilSafeCall(hipFree((char *)cuda_two_to_phi));
            nb_cudafree++;
            cuda_two_to_phi = NULL;
            cutilSafeCall(hipFree((char *)cuda_two_to_minusphi));
            nb_cudafree++;
            cuda_two_to_minusphi = NULL;
        }
        cutilSafeCall(hipFree((char *)g_carry));
        nb_cudafree++;
        g_carry = NULL;
        cutilSafeCall(hipFree((char *)g_err));
        nb_cudafree++;
        g_err = NULL;
        cutilSafeCall(hipFree((char *)g_hlimit));
        nb_cudafree++;
        g_hlimit = NULL;
        cutilSafeCall(hipFree((char *)g_invlimit));
        nb_cudafree++;
        g_invlimit = NULL;
        cutilSafeCall(hipFree((char *)g_limitbv));
        nb_cudafree++;
        g_limitbv = NULL;
        cutilSafeCall(hipFree((char *)cuda_tmp));
        nb_cudafree++;
        cuda_tmp = NULL;
        cufftSafeCall(hipfftDestroy(cuda_fwpx));
        if (generic) {
            cutilSafeCall(hipFree((char *)cuda_tmp_g));
            nb_cudafree++;
            cuda_tmp_g = NULL;
            cutilSafeCall(hipFree((char *)cuda_m));
            nb_cudafree++;
            cuda_m = NULL;
            cutilSafeCall(hipFree((char *)cuda_r));
            nb_cudafree++;
            cuda_r = NULL;
            cutilSafeCall(hipFree((char *)cuda_cm));
            nb_cudafree++;
            cuda_cm = NULL;
            cutilSafeCall(hipFree((char *)cuda_cr));
            nb_cudafree++;
            cuda_cr = NULL;
        }
    }                       // End !cufftonly
    if (compl2) {           // Complex DWT
	gwypfree (cnp);
        nb_free++;
	gwypfree (snp);
        nb_free++;
	gwypfree (cxin);
        nb_free++;
	gwypfree (cyin);
        nb_free++;
        cutilSafeCall(hipFree((char *)cuda_cxin));
        nb_cudafree++;
        cuda_cxin=NULL;
        cutilSafeCall(hipFree((char *)cuda_cyin));
        nb_cudafree++;
        cuda_cyin=NULL;
        cutilSafeCall(hipFree((char *)cuda_cxout));
        nb_cudafree++;
        cuda_cxout=NULL;
        cutilSafeCall(hipFree((char *)cuda_cyout));
        nb_cudafree++;
        cuda_cyout=NULL;
        if (!cufftonly) {
            cutilSafeCall(hipFree((char *)cuda_cnp));
            nb_cudafree++;
            cuda_cnp = NULL;
            cutilSafeCall(hipFree((char *)cuda_snp));
            nb_cudafree++;
            cuda_snp = NULL;
            cutilSafeCall(hipFree((char *)cuda_x));
            nb_cudafree++;
            cuda_x = NULL;
            cutilSafeCall(hipFree((char *)cuda_y));
            nb_cudafree++;
            cuda_y = NULL;
        }
    }                       // End Complex DWT
    else {                  // Real DWT
	gwypfree (xin);    // 12/01/21
        nb_free++;
	gwypfree (yin);    // 12/01/21
        nb_free++;
        cutilSafeCall(hipFree((char *)cuda_xin));
        nb_cudafree++;
        cuda_xin=NULL;
        cutilSafeCall(hipFree((char *)cuda_yin));
        nb_cudafree++;
        cuda_yin=NULL;
        cutilSafeCall(hipFree((char *)cuda_cxout));
        nb_cudafree++;
        cuda_cxout=NULL;
        cutilSafeCall(hipFree((char *)cuda_cyout));
        nb_cudafree++;
        cuda_cyout=NULL;
        if (!cufftonly) {   // 27/01/21
            cutilSafeCall(hipFree((char *)cuda_x));
            nb_cudafree++;
            cuda_x = NULL;
            cutilSafeCall(hipFree((char *)cuda_y));
            nb_cudafree++;
            cuda_y = NULL;
            cutilSafeCall(hipFree((char *)cuda_cxin));
            nb_cudafree++;
            cuda_cxin=NULL;
            cutilSafeCall(hipFree((char *)cuda_cyin));
            nb_cudafree++;
            cuda_cyin=NULL;
        }
        cufftSafeCall(hipfftDestroy(cuda_bwpx));
    }                       // End Real DWT

	if (debug) {
            if (MAXERR != 0.0) {
                sprintf (gwypbuf, "Maximum Round off : %10.10f\n", MAXERR);
                if (printfunction != NULL)
                    (*printfunction)(gwypbuf);
            }
        }
	if (tdebug) {
            gwypwrite_timer (timebuf, 6, TIMER_CLR);
            sprintf (gwypbuf, "CUFFT time : %s\n", timebuf); 
            if (printfunction != NULL)
                (*printfunction)(gwypbuf);
            gwypwrite_timer (timebuf, 5, TIMER_CLR);
            sprintf (gwypbuf, "Squaring and/or Mult. time : %s\n", timebuf); 
            if (printfunction != NULL)
                (*printfunction)(gwypbuf);
            if (cufftonly) {
                gwypwrite_timer (timebuf, 2, TIMER_CLR);
                sprintf (gwypbuf, "Normalization time : %s\n", timebuf); 
                if (printfunction != NULL)
                    (*printfunction)(gwypbuf);
            }
            if (zp) {
                gwypwrite_timer (timebuf, 3, TIMER_CLR); 
                sprintf (gwypbuf, "Modular reduction time : %s\n", timebuf); 
                if (printfunction != NULL)
                    (*printfunction)(gwypbuf);
            }
            if (generic && cufftonly) {
                gwypwrite_timer (timebuf, 4, TIMER_CLR); 
                sprintf (gwypbuf, "Generic copyzero + setzero time : %s\n", timebuf); 
                if (printfunction != NULL)
                    (*printfunction)(gwypbuf);
            }
	}
	if (nb_malloc != nb_free) {
            sprintf (gwypbuf, "Number of CPU mallocs = %d although number of CPU frees = %d\n", nb_malloc, nb_free);
            if (printfunction != NULL)
                (*printfunction)(gwypbuf);
        }
	if (nb_cudamalloc != nb_cudafree) {
            sprintf (gwypbuf, "Number of GPU mallocs = %d although number of GPU frees = %d\n", nb_cudamalloc, nb_cudafree);
            if (printfunction != NULL)
                (*printfunction)(gwypbuf);
        }
        gwypfree (GWP_RANDOM);
        g_fftlen = 0;
        generic = FALSE;
        zp = FALSE;
        setup = FALSE;
        nb_malloc = 0, nb_free = 0, nb_cudamalloc = 0, nb_cudafree = 0;
}
